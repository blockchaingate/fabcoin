#include "hip/hip_runtime.h"
/*
 * Equihash solver created by djeZo (l33tsoftw@gmail.com) for NiceHash
 * Adapted to be more compatible with older C++ compilers
 *
 * cuda_djezo solver was released by NiceHash (www.nicehash.com) under
 * GPL 3.0 license. If you don't have a copy, you can obtain one from
 * https://www.gnu.org/licenses/gpl-3.0.txt
 *
 * Based on CUDA solver by John Tromp released under MIT license.
 * Some helper functions taken out of OpenCL solver by Marc Bevand
 * released under MIT license.
 *
 * Copyright (c) 2016 John Tromp, Marc Bevand
 * Copyright (c) 2017 djeZo, Tanguy Pruvot (GPL v3)
 */
#ifdef _WIN32
#include <Windows.h>
#endif

#include <stdio.h>
#include <vector>

#include "blake2/blake2.h"
#include "eqcuda.hpp"

#define WN 184
#define WK 7
#ifndef MAX_GPUS
#define MAX_GPUS 16
#endif

#define NDIGITS		(WK+1)
#define DIGITBITS	(WN/(NDIGITS))
#define PROOFSIZE 	(1<<WK)
#define BASE 		(1<<DIGITBITS)
#define NHASHES 	(2*BASE)
#define HASHESPERBLAKE (512/WN)
#define HASHOUT 	(HASHESPERBLAKE*WN/8)
#define NBLOCKS 	((NHASHES + HASHESPERBLAKE - 1) / HASHESPERBLAKE)
#define BUCKBITS 	(DIGITBITS - RB)
#define NBUCKETS 	(1 << BUCKBITS)
#define BUCKMASK 	(NBUCKETS - 1)
#define SLOTBITS 	(RB + 2)
#define SLOTRANGE 	(1 << SLOTBITS)
#define NSLOTS SM
#define SLOTMASK 	(SLOTRANGE - 1)
#define NRESTS 		(1 << RB)
#define RESTMASK 	(NRESTS - 1)
#define CANTORBITS 	(2 * SLOTBITS - 2)
#define CANTORMASK 	((1 << CANTORBITS) - 1)
#define CANTORMAXSQRT (2 * NSLOTS)

#define RB8_NSLOTS      640
#define RB8_NSLOTS_LD   624

#define RB_NSLOTS       SLOTRANGE
#define RB_NSLOTS_LD    SLOTRANGE

#define FD_THREADS      128

#ifdef __INTELLISENSE__
// reduce vstudio editor warnings
#include <hip/device_functions.h>
#include <>
#define __launch_bounds__(max_tpb, min_blocks)
#define __CUDA_ARCH__ 520
uint32_t __byte_perm(uint32_t x, uint32_t y, uint32_t z);
uint32_t __byte_perm(uint32_t x, uint32_t y, uint32_t z);
uint32_t __shfl2(uint32_t x, uint32_t y);
uint32_t __shfl_sync(uint32_t mask, uint32_t x, uint32_t y);
uint32_t atomicExch(uint32_t *x, uint32_t y);
uint32_t atomicAdd(uint32_t *x, uint32_t y);
void __syncthreads(void);
void __threadfence(void);
void __threadfence_block(void);
uint32_t __ldg(const uint32_t* address);
uint64_t __ldg(const uint64_t* address);
uint4 __ldca(const uint4 *ptr);
u32 __ldca(const u32 *ptr);
u32 umin(const u32, const u32);
u32 umax(const u32, const u32);
#endif

#define OPT_SYNC_ALL

#if CUDA_VERSION >= 9000 && __CUDA_ARCH__ >= 300
#define __shfl2(var, srcLane)  __shfl_sync(0xFFFFFFFFu, var, srcLane)
#undef __any
#define __any(p) __any_sync(0xFFFFFFFFu, p)
#else
#define __shfl2 __shfl
#endif

struct __align__(32) slot {
	u32 hash[8];
};

struct __align__(16) slotsmall {
	u32 hash[4];
};

struct __align__(8) slottiny {
	u32 hash[2];
};

struct __align__(16) slotmix{
	slotsmall treessmall[128];
	slottiny treestiny[128];
};

typedef slot		bucket[128];
typedef slotsmall	bucketsmall[128];
typedef slottiny	buckettiny[128];

template <u32 RB, u32 SM>
struct equi
{
	bucket		*round0trees;
	bucket		*round1trees;
	bucket		*round2trees;
	bucket		*round3trees;
	bucketsmall	*round4trees;
	bucketsmall	*round5trees;
	buckettiny	*round6trees;
    void        *reserved;
    union {
        u64 blake_h[8];
        u32 blake_h32[16];
    };
    struct {
		u32 nslots[7][NBUCKETS];
        u32 nonce[4];
	} edata;
	scontainerreal srealcont;
};

#define byteswap32(x) \
    ( (((x) & 0xff000000) >> 24) | (((x) & 0x00ff0000) >> 8) | \
    (((x) & 0x0000ff00) << 8) | (((x) & 0x000000ff) << 24) )


// todo: use cuda_helper.h and/or cuda_vector.h
__device__ __forceinline__ uint2 operator^ (uint2 a, uint2 b)
{
	return make_uint2(a.x ^ b.x, a.y ^ b.y);
}

__device__ __forceinline__ uint4 operator^ (uint4 a, uint4 b)
{
	return make_uint4(a.x ^ b.x, a.y ^ b.y, a.z ^ b.z, a.w ^ b.w);
}

// for ROR 63 (or ROL 1); this func only support (32 <= offset < 64)
__device__ __forceinline__ uint2 ROR2(const uint2 a, const int offset)
{
	uint2 result;
#if __CUDA_ARCH__ > 300
	{
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
	}
#else
	result.y = ((a.x >> (offset - 32)) | (a.y << (64 - offset)));
	result.x = ((a.y >> (offset - 32)) | (a.x << (64 - offset)));
#endif
	return result;
}


__device__ __forceinline__ uint2 SWAPUINT2(uint2 value)
{
	return make_uint2(value.y, value.x);
}

__device__ __forceinline__ uint2 ROR24(const uint2 a)
{
	uint2 result;
	result.x = __byte_perm(a.y, a.x, 0x2107);
	result.y = __byte_perm(a.y, a.x, 0x6543);
	return result;
}

__device__ __forceinline__ uint2 ROR16(const uint2 a)
{
	uint2 result;
	result.x = __byte_perm(a.y, a.x, 0x1076);
	result.y = __byte_perm(a.y, a.x, 0x5432);
	return result;
}

__device__ __forceinline__ void G2(u64 & a, u64 & b, u64 & c, u64 & d, u64 x, u64 y)
{
	a = a + b + x;
	((uint2*)&d)[0] = SWAPUINT2(((uint2*)&d)[0] ^ ((uint2*)&a)[0]);
	c = c + d;
	((uint2*)&b)[0] = ROR24(((uint2*)&b)[0] ^ ((uint2*)&c)[0]);
	a = a + b + y;
	((uint2*)&d)[0] = ROR16(((uint2*)&d)[0] ^ ((uint2*)&a)[0]);
	c = c + d;
	((uint2*)&b)[0] = ROR2(((uint2*)&b)[0] ^ ((uint2*)&c)[0], 63U);
}

// untested..
struct packer_default
{
	__device__ __forceinline__ static u32 set_bucketid_and_slots(const u32 bucketid, const u32 s0, const u32 s1, const u32 RB)
	{
		return (((bucketid << SLOTBITS) | s0) << SLOTBITS) | s1;
	}

	__device__ __forceinline__ static u32 get_bucketid(const u32 bid, const u32 RB, const u32 SM)
	{
		// BUCKMASK-ed to prevent illegal memory accesses in case of memory errors
		return (bid >> (2 * SLOTBITS)) & BUCKMASK;
	}

	__device__ __forceinline__ static u32 get_slot0(const u32 bid, const u32 s1, const u32 RB, const u32 SM)
	{
		return bid & SLOTMASK;
	}

	__device__ __forceinline__ static u32 get_slot1(const u32 bid, const u32 RB, const u32 SM)
	{
		return (bid >> SLOTBITS) & SLOTMASK;
	}
};


struct packer_cantor
{
	__device__ __forceinline__ static u32 cantor(const u32 s0, const u32 s1)
	{
		u32 a = umax(s0, s1);
		u32 b = umin(s0, s1);
		return a * (a + 1) / 2 + b;
	}

	__device__ __forceinline__ static u32 set_bucketid_and_slots(const u32 bucketid, const u32 s0, const u32 s1, const u32 RB)
	{
		return (bucketid << CANTORBITS) | cantor(s0, s1);
	}

	__device__ __forceinline__ static u32 get_bucketid(const u32 bid, const u32 RB, const u32 SM)
	{
		return (bid >> CANTORBITS) & BUCKMASK;
	}

	__device__ __forceinline__ static u32 get_slot0(const u32 bid, const u32 s1, const u32 RB, const u32 SM)
	{
		return ((bid & CANTORMASK) - cantor(0, s1)) & SLOTMASK;
	}

	__device__ __forceinline__ static u32 get_slot1(const u32 bid, const u32 RB, const u32 SM)
	{
		u32 k, q, sqr = 8 * (bid & CANTORMASK) + 1;
		// this k=sqrt(sqr) computing loop averages 3.4 iterations out of maximum 9
		for (k = CANTORMAXSQRT; (q = sqr / k) < k; k = (k + q) / 2);
		return ((k - 1) / 2) & SLOTMASK;
	}
};

__device__ __constant__ const u64 blake_iv[] = {
	0x6a09e667f3bcc908, 0xbb67ae8584caa73b,
	0x3c6ef372fe94f82b, 0xa54ff53a5f1d36f1,
	0x510e527fade682d1, 0x9b05688c2b3e6c1f,
	0x1f83d9abfb41bd6b, 0x5be0cd19137e2179,
};

__device__ __constant__ const u8 blake2b_sigma[12][16] =
{
    {  0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 }, // 1
    { 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 }, // 2
    { 11,  8, 12,  0,  5,  2, 15, 13, 10, 14,  3,  6,  7,  1,  9,  4 }, // 3
    {  7,  9,  3,  1, 13, 12, 11, 14,  2,  6,  5, 10,  4,  0, 15,  8 }, // 4
    {  9,  0,  5,  7,  2,  4, 10, 15, 14,  1, 11, 12,  6,  8,  3, 13 }, // 5
    {  2, 12,  6, 10,  0, 11,  8,  3,  4, 13,  7,  5, 15, 14,  1,  9 }, // 6
    { 12,  5,  1, 15, 14, 13,  4, 10,  0,  7,  6,  3,  9,  2,  8, 11 }, // 7
    { 13, 11,  7, 14, 12,  1,  3,  9,  5,  0, 15,  4,  8,  6,  2, 10 }, // 8
    {  6, 15, 14,  9, 11,  3,  0,  8, 12,  2, 13,  7,  1,  4, 10,  5 }, // 9
    { 10,  2,  8,  4,  7,  6,  1,  5, 15, 11,  9, 14,  3, 12, 13,  0 }, // 10
    {  0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 }, // 11
    { 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 }, // 12
};


#if CUDART_VERSION < 8000 || !defined(__ldca)
#define __ldca(ptr) *(ptr)
#endif

template <u32 RB, u32 SM, typename PACKER>
__global__ void digit_first(equi<RB, SM>* eq, u8 *buf, const u32 nonce)
{
	const u32 block = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ u64 hash_h[8];

	u32* hash_h32 = (u32*)hash_h;

	if (threadIdx.x < 16)
	{
		hash_h32[threadIdx.x] = __ldca(&eq->blake_h32[threadIdx.x]);
    }
    
	if ( block == 0 )
		eq->edata.nonce[0] = nonce;	

	__syncthreads();
	
    u64 val1 = 0, val9 = 0;	
	u8 len = 76;
	u64 tmpbuf[16];

	*(ulonglong4 *)(tmpbuf + 0) = *(ulonglong4 *)(buf + 0);
	*(ulonglong4 *)(tmpbuf + 4) = *(ulonglong4 *)(buf + 32);
	*(ulonglong4 *)(tmpbuf + 8) = *(ulonglong4 *)(buf + 64);
	*((u32 *)tmpbuf + 14) = nonce;

	val1 = tmpbuf[1];
    val9 = tmpbuf[9] | (u64)block<<(len%8*8);

	union
	{
		u64 v[16];
		u32 v32[32];
		uint4 v128[8];
	};

	v[0] = hash_h[0];
	v[1] = hash_h[1];
	v[2] = hash_h[2];
	v[3] = hash_h[3];
	v[4] = hash_h[4];
	v[5] = hash_h[5];
	v[6] = hash_h[6];
	v[7] = hash_h[7];
	v[8] = blake_iv[0];
	v[9] = blake_iv[1];
	v[10] = blake_iv[2];
	v[11] = blake_iv[3];
	v[12] = blake_iv[4] ^ (128 + len + 4);
	v[13] = blake_iv[5];
	v[14] = blake_iv[6] ^ 0xffffffffffffffff;
	v[15] = blake_iv[7];

	// mix 1
    const u8  *s = blake2b_sigma[0];
	G2(v[0], v[4], v[8], v[12], tmpbuf[s[0]], val1 );
	G2(v[1], v[5], v[9], v[13], tmpbuf[s[2]], tmpbuf[s[3]]);
	G2(v[2], v[6], v[10], v[14], tmpbuf[s[4]], tmpbuf[s[5]]);
	G2(v[3], v[7], v[11], v[15], tmpbuf[s[6]], tmpbuf[s[7]]);
	G2(v[0], v[5], v[10], v[15], tmpbuf[s[8]], val9);
	G2(v[1], v[6], v[11], v[12], tmpbuf[s[10]], tmpbuf[s[11]]);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 2
	s = blake2b_sigma[1];
	G2(v[0], v[4], v[8], v[12], 0, tmpbuf[s[1]]);
	G2(v[1], v[5], v[9], v[13], tmpbuf[s[2]], tmpbuf[s[3]]);
	G2(v[2], v[6], v[10], v[14], val9, 0);
	G2(v[3], v[7], v[11], v[15], 0, tmpbuf[s[7]]);
	G2(v[0], v[5], v[10], v[15], val1, 0);
	G2(v[1], v[6], v[11], v[12], tmpbuf[s[10]], tmpbuf[s[11]]);
	G2(v[2], v[7], v[8], v[13], tmpbuf[s[12]], tmpbuf[s[13]]);
	G2(v[3], v[4], v[9], v[14], tmpbuf[s[14]], tmpbuf[s[15]]);

	// mix 3
	s = blake2b_sigma[2];
	G2(v[0], v[4], v[8], v[12], tmpbuf[s[0]], tmpbuf[s[1]]);
	G2(v[1], v[5], v[9], v[13], 0, tmpbuf[s[3]]);
	G2(v[2], v[6], v[10], v[14], tmpbuf[s[4]], tmpbuf[s[5]]);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], tmpbuf[s[8]], 0);
	G2(v[1], v[6], v[11], v[12], tmpbuf[s[10]], tmpbuf[s[11]]);
	G2(v[2], v[7], v[8], v[13], tmpbuf[s[12]], val1);
	G2(v[3], v[4], v[9], v[14], val9, tmpbuf[s[15]]);

	// mix 4
	s = blake2b_sigma[3];
	G2(v[0], v[4], v[8], v[12], tmpbuf[s[0]], val9);
	G2(v[1], v[5], v[9], v[13], tmpbuf[s[2]], val1);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], tmpbuf[s[6]], 0);
	G2(v[0], v[5], v[10], v[15], tmpbuf[s[8]], tmpbuf[s[9]]);
	G2(v[1], v[6], v[11], v[12], tmpbuf[s[10]], tmpbuf[s[11]]);
	G2(v[2], v[7], v[8], v[13], tmpbuf[s[12]], tmpbuf[s[13]]);
	G2(v[3], v[4], v[9], v[14], 0, tmpbuf[s[15]]); 

	// mix 5
	s = blake2b_sigma[4];
	G2(v[0], v[4], v[8], v[12], val9, tmpbuf[s[1]]);
	G2(v[1], v[5], v[9], v[13], tmpbuf[s[2]], tmpbuf[s[3]]);
	G2(v[2], v[6], v[10], v[14], tmpbuf[s[4]], tmpbuf[s[5]]);
	G2(v[3], v[7], v[11], v[15], tmpbuf[s[6]], 0);
	G2(v[0], v[5], v[10], v[15], 0, val1);
	G2(v[1], v[6], v[11], v[12], tmpbuf[s[10]], 0);
	G2(v[2], v[7], v[8], v[13], tmpbuf[s[12]], tmpbuf[s[13]]);
	G2(v[3], v[4], v[9], v[14], tmpbuf[s[14]], 0);

	// mix 6
	s = blake2b_sigma[5];
	G2(v[0], v[4], v[8], v[12], tmpbuf[s[0]], 0);
	G2(v[1], v[5], v[9], v[13], tmpbuf[s[2]], tmpbuf[s[3]]);
	G2(v[2], v[6], v[10], v[14], tmpbuf[s[4]], tmpbuf[s[5]]);
	G2(v[3], v[7], v[11], v[15], tmpbuf[s[6]], tmpbuf[s[7]]);
	G2(v[0], v[5], v[10], v[15], tmpbuf[s[8]], 0);
	G2(v[1], v[6], v[11], v[12], tmpbuf[s[10]], tmpbuf[s[11]]);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], val1, val9);

	// mix 7
	s = blake2b_sigma[6];
	G2(v[0], v[4], v[8], v[12], 0, tmpbuf[s[1]]);
	G2(v[1], v[5], v[9], v[13], val1, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], tmpbuf[s[6]], tmpbuf[s[7]]);
	G2(v[0], v[5], v[10], v[15], tmpbuf[s[8]], tmpbuf[s[9]]);
	G2(v[1], v[6], v[11], v[12], tmpbuf[s[10]], tmpbuf[s[11]]);
	G2(v[2], v[7], v[8], v[13], val9, tmpbuf[s[13]]);
	G2(v[3], v[4], v[9], v[14], tmpbuf[s[14]], tmpbuf[s[15]]);

	// mix 8
	s = blake2b_sigma[7];
	G2(v[0], v[4], v[8], v[12], 0, tmpbuf[s[1]]);
	G2(v[1], v[5], v[9], v[13], tmpbuf[s[2]], 0);
	G2(v[2], v[6], v[10], v[14], 0, val1);
	G2(v[3], v[7], v[11], v[15], tmpbuf[s[6]], val9);
	G2(v[0], v[5], v[10], v[15], tmpbuf[s[8]], tmpbuf[s[9]]);
	G2(v[1], v[6], v[11], v[12], 0, tmpbuf[s[11]]);
	G2(v[2], v[7], v[8], v[13], tmpbuf[s[12]], tmpbuf[s[13]]);
	G2(v[3], v[4], v[9], v[14], tmpbuf[s[14]], tmpbuf[s[15]]);

	// mix 9
	s = blake2b_sigma[8];
	G2(v[0], v[4], v[8], v[12], tmpbuf[s[0]], 0);
	G2(v[1], v[5], v[9], v[13], 0, val9);
	G2(v[2], v[6], v[10], v[14], tmpbuf[s[4]], tmpbuf[s[5]]);
	G2(v[3], v[7], v[11], v[15], tmpbuf[s[6]], tmpbuf[s[7]]);
	G2(v[0], v[5], v[10], v[15], 0, tmpbuf[s[9]]);
	G2(v[1], v[6], v[11], v[12], 0, tmpbuf[s[11]]);
	G2(v[2], v[7], v[8], v[13], val1, tmpbuf[s[13]]);
	G2(v[3], v[4], v[9], v[14], tmpbuf[s[14]], tmpbuf[s[15]]);

	// mix 10
	s = blake2b_sigma[9];
	G2(v[0], v[4], v[8], v[12], tmpbuf[s[0]], tmpbuf[s[1]]);
	G2(v[1], v[5], v[9], v[13], tmpbuf[s[2]], tmpbuf[s[3]]);
	G2(v[2], v[6], v[10], v[14], tmpbuf[s[4]], tmpbuf[s[5]]);
	G2(v[3], v[7], v[11], v[15], val1, tmpbuf[s[7]]);
	G2(v[0], v[5], v[10], v[15], 0, tmpbuf[s[9]]);
	G2(v[1], v[6], v[11], v[12], val9, 0);
	G2(v[2], v[7], v[8], v[13], tmpbuf[s[12]], 0);
	G2(v[3], v[4], v[9], v[14], 0, tmpbuf[s[15]]);

	// mix 11
	s = blake2b_sigma[10];
	G2(v[0], v[4], v[8], v[12], tmpbuf[s[0]], val1);
	G2(v[1], v[5], v[9], v[13], tmpbuf[s[2]], tmpbuf[s[3]]);
	G2(v[2], v[6], v[10], v[14], tmpbuf[s[4]], tmpbuf[s[5]]);
	G2(v[3], v[7], v[11], v[15], tmpbuf[s[6]], tmpbuf[s[7]]);
	G2(v[0], v[5], v[10], v[15], tmpbuf[s[8]], val9);
	G2(v[1], v[6], v[11], v[12], tmpbuf[s[10]], tmpbuf[s[11]]);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 12
	s = blake2b_sigma[11];
	G2(v[0], v[4], v[8], v[12], 0, tmpbuf[s[1]]);
	G2(v[1], v[5], v[9], v[13], tmpbuf[s[2]], tmpbuf[s[3]]);
	G2(v[2], v[6], v[10], v[14], val9, 0);
	G2(v[3], v[7], v[11], v[15], 0, tmpbuf[s[7]]);
	G2(v[0], v[5], v[10], v[15], val1, 0);
	G2(v[1], v[6], v[11], v[12], tmpbuf[s[10]], tmpbuf[s[11]]);
	G2(v[2], v[7], v[8], v[13], tmpbuf[s[12]], tmpbuf[s[13]]);
	G2(v[3], v[4], v[9], v[14], tmpbuf[s[14]], tmpbuf[s[15]]);

	v[0] ^= hash_h[0] ^ v[8];
	v[1] ^= hash_h[1] ^ v[9];
	v[2] ^= hash_h[2] ^ v[10];
	v[3] ^= hash_h[3] ^ v[11];
	v[4] ^= hash_h[4] ^ v[12];
	v[5] ^= hash_h[5] ^ v[13];
    v[6] ^= hash_h[6] ^ v[14];
    v[7] ^= hash_h[7] ^ v[15];

	u32 bucketid;
    u32 bexor = __byte_perm(v32[0], 0, 0x4012); // first 20 bits
    asm("bfe.u32 %0, %1, 6, 18;" : "=r"(bucketid) : "r"(bexor));
	u32 slotp = atomicAdd(&eq->edata.nslots[0][bucketid], 1);
	if (slotp < RB_NSLOTS)
	{
		slot* s = &eq->round0trees[bucketid][slotp];

		ulonglong4 tt;
		tt.x = __byte_perm(v32[0], v32[1], 0x2345) | ((u64)__byte_perm(v32[1], v32[2], 0x2345) << 32);
		tt.y = __byte_perm(v32[2], v32[3], 0x2345) | ((u64)__byte_perm(v32[3], v32[4], 0x2345) << 32);
		tt.z = __byte_perm(v32[4], v32[5], 0x2345) | ((u64)__byte_perm(v32[5], v32[6], 0x2345) << 32);
		tt.w = (u64)block << 33;

		*(ulonglong4*)(&s->hash[0]) = tt;
	}

    bexor = __byte_perm(v32[5], v32[6], 0x3456);
    asm("bfe.u32 %0, %1, 14, 18;" : "=r"(bucketid) : "r"(bexor));

    slotp = atomicAdd(&eq->edata.nslots[0][bucketid], 1);
	if (slotp < RB_NSLOTS)
	{
		slot* s = &eq->round0trees[bucketid][slotp];

		ulonglong4 tt;
		tt.x = __byte_perm(v32[6], v32[7], 0x1234) | ((u64)__byte_perm(v32[7], v32[8], 0x1234)<<32);
		tt.y = __byte_perm(v32[8], v32[9], 0x1234) | ((u64)__byte_perm(v32[9], v32[10], 0x1234)<<32);
		tt.z = __byte_perm(v32[10], v32[11], 0x1234) | ((u64)__byte_perm(v32[11], v32[12], 0x1234)<<32);
		tt.w = (((u64)block << 1) + 1)<<32;

		*(ulonglong4*)(&s->hash[0]) = tt;
	}
}

/*
  Functions digit_1 to digit_8 works by the same principle;
  Each thread does 2-3 slot loads (loads are coalesced).
  Xorwork of slots is loaded into shared memory and is kept in registers (except for digit_1).
  At the same time, restbits (8 or 9 bits) in xorwork are used for collisions.
  Restbits determine position in ht.
  Following next is pair creation. First one (or two) pairs' xorworks are put into global memory
  as soon as possible, the rest pairs are saved in shared memory (one u32 per pair - 16 bit indices).
  In most cases, all threads have one (or two) pairs so with this trick, we offload memory writes a bit in last step.
  In last step we save xorwork of pairs in memory.
*/
template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS, u32 THREADS>
__global__ void digit_1(equi<RB, SM>* eq)
{
	__shared__ u16 ht[THREADS][SSM - 1];
	__shared__ uint2 lastword1[RB_NSLOTS];
	__shared__ uint4 lastword2[RB_NSLOTS];
	__shared__ int ht_len[MAXPAIRS];
	__shared__ u32 pairs_len;
	__shared__ u32 next_pair;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;   

	// reset hashtable len
	if (threadid < NRESTS)
    {
		ht_len[threadid] = 0;
    }

    pairs_len = 0;
    next_pair = 0;    

	u32 bsize = umin(eq->edata.nslots[0][bucketid], RB_NSLOTS);

	u32 hr[2];
	int pos[2];
	pos[0] = pos[1] = SSM;

	uint2 ta[2];
	uint4 tb[2];

	u32 si[2];

#ifdef OPT_SYNC_ALL
	// enable this to make fully safe shared mem operations;
	// disabled gains some speed, but can rarely cause a crash
	__syncthreads();
#endif
#pragma unroll
	for (u32 i = 0; i < 2; ++i)
	{
		si[i] = i * THREADS + threadid;
		if (si[i] >= bsize) break;

        const slot* pslot1 = eq->round0trees[bucketid] + si[i];
		// get xhash
		uint4 a1 = *(uint4*)(&pslot1->hash[0]);
		uint2 a2 = *(uint2*)(&pslot1->hash[4]);
		ta[i].x = a1.x;
		ta[i].y = a1.y;
		lastword1[si[i]] = ta[i];
		tb[i].x = a1.z;
		tb[i].y = a1.w;
		tb[i].z = a2.x;
		tb[i].w = a2.y;
		lastword2[si[i]] = tb[i];

        asm("bfe.u32 %0, %1, 25, 5;" : "=r"(hr[i]) : "r"(ta[i].x));

        pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) 
            ht[hr[i]][pos[i]] = si[i];
	}
	__syncthreads();
	int* pairs = ht_len;

	u32 xors[6];
	u32 xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i < 2; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u16 p = ht[hr[i]][0];

			*(uint2*)(&xors[0]) = ta[i] ^ lastword1[p];

            asm("bfe.u32 %0, %1, 7, %2;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(BUCKBITS));

            xorslot = atomicAdd(&eq->edata.nslots[1][xorbucketid], 1);

			if (xorslot < NSLOTS)
			{
				*(uint4*)(&xors[2]) = lastword2[si[i]] ^ lastword2[p];

				slot &xs = eq->round1trees[xorbucketid][xorslot];
                		ulonglong4 ttx;
				ttx.x = __byte_perm(xors[0], xors[1], 0x0765) | ((u64)__byte_perm(xors[1], xors[2], 0x0765)<<32);
				ttx.y = __byte_perm(xors[2], xors[3], 0x0765) | ((u64)__byte_perm(xors[3], xors[4], 0x0765)<<32);
				ttx.z = __byte_perm(xors[4], xors[5], 0x0765) | ((u64)PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB)<<32);
				*(ulonglong4*)(&xs.hash[0]) = ttx;
			}

			for (int k = 1; k < pos[i]; ++k)
			{
				u32 pindex = atomicAdd(&pairs_len, 1);
				if (pindex >= MAXPAIRS) break;
				u16 prev = ht[hr[i]][k];
                pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
			}
		}
	}

	__syncthreads();

	// process pairs
	u32 plen = umin(pairs_len, MAXPAIRS);

	u32 i, k;
	for (u32 s = atomicAdd(&next_pair, 1); s < plen; s = atomicAdd(&next_pair, 1))
	{
		int pair = pairs[s];
		i = __byte_perm(pair, 0, 0x4510);
		k = __byte_perm(pair, 0, 0x4532);

		*(uint2*)(&xors[0]) = lastword1[i] ^ lastword1[k];

        asm("bfe.u32 %0, %1, 7, %2;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(BUCKBITS));
		
        xorslot = atomicAdd(&eq->edata.nslots[1][xorbucketid], 1);

		if (xorslot < NSLOTS)
		{
			*(uint4*)(&xors[2]) = lastword2[i] ^ lastword2[k];

			slot &xs = eq->round1trees[xorbucketid][xorslot];
			ulonglong4 ttx;
			ttx.x = __byte_perm(xors[0], xors[1], 0x0765) | ((u64)__byte_perm(xors[1], xors[2], 0x0765) << 32);
			ttx.y = __byte_perm(xors[2], xors[3], 0x0765) | ((u64)__byte_perm(xors[3], xors[4], 0x0765) << 32);
			ttx.z = __byte_perm(xors[4], xors[5], 0x0765) | ((u64)PACKER::set_bucketid_and_slots(bucketid, i, k, RB) << 32);
			*(ulonglong4*)(&xs.hash[0]) = ttx;
		}
	}
}

template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS, u32 THREADS>
__global__ void digit_2(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][SSM - 1];
	__shared__ uint4 lastword1[NSLOTS];
	__shared__ uint2 lastword2[NSLOTS];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;
	__shared__ u32 next_pair;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	if (threadid < NRESTS)
		ht_len[threadid] = 0;
	else if (threadid == (THREADS - 1))
		pairs_len = 0;
	else if (threadid == (THREADS - 33))
		next_pair = 0;

    pairs_len = 0;
    next_pair = 0;    

	u32 bsize = umin(eq->edata.nslots[1][bucketid], NSLOTS);

	u32 hr[2];
	int pos[2];
	pos[0] = pos[1] = SSM;

	uint4 ta[2];
	uint2 tb[2];

	u32 si[2];
#ifdef OPT_SYNC_ALL
	__syncthreads();
#endif
#pragma unroll 2
	for (u32 i = 0; i < 2; i++)
	{
		si[i] = i * THREADS + threadid;
		if (si[i] >= bsize) break;

        slot &xs = eq->round1trees[bucketid][si[i]];

        ta[i] = *(uint4*)(&xs.hash[0]);
        lastword1[si[i]] = ta[i];

        tb[i] = *(uint2*)(&xs.hash[4]);
        lastword2[si[i]] = tb[i];

        asm("bfe.u32 %0, %1, 26, %2;" : "=r"(hr[i]) : "r"(ta[i].x), "r"(RB));
        pos[i] = atomicAdd(&ht_len[hr[i]], 1);
        if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

    __syncthreads();

	u32 xors[5];
	u32 xorbucketid, xorslot;

#pragma unroll 2
	for (u32 i = 0; i < 2; i++)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u16 p = ht[hr[i]][0];

            *(uint4*)(&xors[0]) = ta[i] ^ lastword1[p];

            asm("bfe.u32 %0, %1, 8, %2;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(BUCKBITS));

            xorslot = atomicAdd(&eq->edata.nslots[2][xorbucketid], 1);
			if (xorslot < NSLOTS)
			{
                		*(uint2*)(&xors[4]) = lastword2[si[i]] ^ lastword2[p];

				slot &xs = eq->round2trees[xorbucketid][xorslot];

				ulonglong4 ttx;
				ttx.x = __byte_perm(xors[0], xors[1], 0x0765) | ((u64)__byte_perm(xors[1], xors[2], 0x0765)<<32);
				ttx.y = __byte_perm(xors[2], xors[3], 0x0765) | ((u64)__byte_perm(xors[3], xors[4], 0x0765)<<32);				
				ttx.z = (u64)PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB)<<32;
				ttx.w = 0;
				*(ulonglong4 *)(&xs.hash[0]) = ttx;

			}

			for (int k = 1; k != pos[i]; ++k)
			{
				u32 pindex = atomicAdd(&pairs_len, 1);
				if (pindex >= MAXPAIRS) break;
				u16 prev = ht[hr[i]][k];
				pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
			}            
		}
	}

    __syncthreads();

	// process pairs
	u32 plen = umin(pairs_len, MAXPAIRS);

	u32 i, k;
	for (u32 s = atomicAdd(&next_pair, 1); s < plen; s = atomicAdd(&next_pair, 1))
	{
		int pair = pairs[s];
		i = __byte_perm(pair, 0, 0x4510);
		k = __byte_perm(pair, 0, 0x4532);
        *(uint4*)(&xors[0]) = lastword1[i] ^ lastword1[k];
        asm("bfe.u32 %0, %1, 8, %2;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(BUCKBITS));

		xorslot = atomicAdd(&eq->edata.nslots[2][xorbucketid], 1);
		if (xorslot < NSLOTS)
		{
            		*(uint2*)(&xors[4]) = lastword2[i] ^ lastword2[k];

			slot &xs = eq->round2trees[xorbucketid][xorslot];

			ulonglong4 ttx;
			ttx.x = __byte_perm(xors[0], xors[1], 0x0765) | ((u64)__byte_perm(xors[1], xors[2], 0x0765)<<32);
			ttx.y = __byte_perm(xors[2], xors[3], 0x0765) | ((u64)__byte_perm(xors[3], xors[4], 0x0765)<<32);
			ttx.z = (u64)PACKER::set_bucketid_and_slots(bucketid, i, k, RB) << 32;
			ttx.w = 0;
			*(ulonglong4 *)(&xs.hash[0]) = ttx;
		}
	}
}


template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS, u32 THREADS>
__global__ void digit_3(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ uint4 lastword1[NSLOTS];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;
	__shared__ u32 next_pair;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	if (threadid < NRESTS)
		ht_len[threadid] = 0;
	else if (threadid == (THREADS - 1))
		pairs_len = 0;
	else if (threadid == (THREADS - 33))
		next_pair = 0;

    pairs_len = 0;
    next_pair = 0;    

	u32 bsize = umin(eq->edata.nslots[2][bucketid], NSLOTS);

	u32 hr[2];
	int pos[2];
	pos[0] = pos[1] = SSM;

	u32 si[2];
	uint4 tt[2];

#ifdef OPT_SYNC_ALL
	__syncthreads();
#endif

#pragma unroll 2
	for (u32 i = 0; i < 2; i++)
	{
		si[i] = i * THREADS + threadid;
		if (si[i] >= bsize) break;
		slot &xs = eq->round2trees[bucketid][si[i]];

		tt[i] = *(uint4*)(&xs.hash[0]);
		lastword1[si[i]] = tt[i];
		asm("bfe.u32 %0, %1, 27, %2;" : "=r"(hr[i]) : "r"(tt[i].x), "r"(RB));
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];        
	}

	__syncthreads();

	u32 xors[5];
	u32 xorbucketid, xorslot;

#pragma unroll 2
	for (u32 i = 0; i < 2; i++)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u16 p = ht[hr[i]][0];

			*(uint4*)(&xors[0]) = tt[i] ^ lastword1[p];

            asm("bfe.u32 %0, %1, 9, %2;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(BUCKBITS));
            
            xorslot = atomicAdd(&eq->edata.nslots[3][xorbucketid], 1);

			if (xorslot < NSLOTS)
			{
				slot &xs = eq->round3trees[xorbucketid][xorslot];

				ulonglong4 ttx;
				ttx.x = __byte_perm(xors[0], xors[1], 0x1076) | ((u64)__byte_perm(xors[1], xors[2], 0x1076)<<32);
				ttx.y = __byte_perm(xors[2], xors[3], 0x1076) | ((u64)__byte_perm(xors[3], xors[4], 0x1076)<<32);
				ttx.z = (u64)PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB) << 32;
				*(ulonglong4 *)(&xs.hash[0]) = ttx;
			}

			for (int k = 1; k != pos[i]; ++k)
			{
				u32 pindex = atomicAdd(&pairs_len, 1);
				if (pindex >= MAXPAIRS) break;
				u16 prev = ht[hr[i]][k];
				pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
			}
		}
	}

	__syncthreads();

	// process pairs
	u32 plen = umin(pairs_len, MAXPAIRS);

	u32 i, k;
	for (u32 s = atomicAdd(&next_pair, 1); s < plen; s = atomicAdd(&next_pair, 1))
	{
		int pair = pairs[s];
		i = __byte_perm(pair, 0, 0x4510);
		k = __byte_perm(pair, 0, 0x4532);

        *(uint4*)(&xors[0]) = lastword1[i] ^ lastword1[k];

        asm("bfe.u32 %0, %1, 9, %2;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(BUCKBITS));
        xorslot = atomicAdd(&eq->edata.nslots[3][xorbucketid], 1);
        
        if (xorslot < NSLOTS)
		{
			slot &xs = eq->round3trees[xorbucketid][xorslot];

			ulonglong4 ttx;
			ttx.x = __byte_perm(xors[0], xors[1], 0x1076) | ((u64)__byte_perm(xors[1], xors[2], 0x1076)<<32);
			ttx.y = __byte_perm(xors[2], xors[3], 0x1076) | ((u64)__byte_perm(xors[3], xors[4], 0x1076)<<32);
			ttx.z = (u64)PACKER::set_bucketid_and_slots(bucketid, i, k, RB) << 32;
			ttx.w = 0;
			*(ulonglong4 *)(&xs.hash[0]) = ttx;
		}		
	}
}

template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS, u32 THREADS>
__global__ void digit_4(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ uint4 lastword[NSLOTS];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;
	__shared__ u32 next_pair;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	if (threadid < NRESTS)
		ht_len[threadid] = 0;
	else if (threadid == (THREADS - 1))
		pairs_len = 0;
	else if (threadid == (THREADS - 33))
		next_pair = 0;

    pairs_len = 0;
    next_pair = 0;

	u32 bsize = umin(eq->edata.nslots[3][bucketid], NSLOTS);

	u32 hr[2];
	int pos[2];
	pos[0] = pos[1] = SSM;

	u32 si[2];
	uint4 tt[2];
#ifdef OPT_SYNC_ALL
	__syncthreads();
#endif
#pragma unroll 2
	for (u32 i = 0; i < 2; i++)
	{
		si[i] = i * THREADS + threadid;
		if (si[i] >= bsize) break;

		slot &xs = eq->round3trees[bucketid][si[i]];

		// get xhash
		tt[i] = *(uint4*)(&xs.hash[0]);
		lastword[si[i]] = tt[i];

        asm("bfe.u32 %0, %1, 20, %2;" : "=r"(hr[i]) : "r"(tt[i].x), "r"(RB));

		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}
   
	__syncthreads();
	u32 xors[4];
	u32 xorbucketid, xorslot;

#pragma unroll 2
	for (u32 i = 0; i < 2; i++)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u16 p = ht[hr[i]][0];

			*(uint4*)(&xors[0]) = tt[i] ^ lastword[p];

			asm("bfe.u32 %0, %1, 2, %2;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(BUCKBITS));
			xorslot = atomicAdd(&eq->edata.nslots[4][xorbucketid], 1);
			if (xorslot < NSLOTS)
			{
				slotsmall &xs = eq->round4trees[xorbucketid][xorslot];

                uint4 ttx;
                ttx.x = __byte_perm(xors[0], xors[1], 0x0765);
                ttx.y = __byte_perm(xors[1], xors[2], 0x0765);
                ttx.z = __byte_perm(xors[2], xors[3], 0x0765);

                ttx.w = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB);
                *(uint4*)(&xs.hash[0]) = ttx;
			}

			for (int k = 1; k != pos[i]; ++k)
			{
				u32 pindex = atomicAdd(&pairs_len, 1);
				if (pindex >= MAXPAIRS) break;
				u16 prev = ht[hr[i]][k];
				pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
			}
		}
	}

	__syncthreads();

	// process pairs
	u32 plen = umin(pairs_len, MAXPAIRS);
	u32 i, k;
	for (u32 s = atomicAdd(&next_pair, 1); s < plen; s = atomicAdd(&next_pair, 1))
	{
		int pair = pairs[s];
		i = __byte_perm(pair, 0, 0x4510);
		k = __byte_perm(pair, 0, 0x4532);

		*(uint4*)(&xors[0]) = lastword[i] ^ lastword[k];

        asm("bfe.u32 %0, %1, 2, %2;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(BUCKBITS));
        xorslot = atomicAdd(&eq->edata.nslots[4][xorbucketid], 1);
        if (xorslot < NSLOTS)
        {
            slotsmall &xs = eq->round4trees[xorbucketid][xorslot];

            uint4 ttx;
            ttx.x = __byte_perm(xors[0], xors[1], 0x0765);
            ttx.y = __byte_perm(xors[1], xors[2], 0x0765);
            ttx.z = __byte_perm(xors[2], xors[3], 0x0765);

            ttx.w = PACKER::set_bucketid_and_slots(bucketid, i, k, RB);
            *(uint4*)(&xs.hash[0]) = ttx;
        }
    }   
}

template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS, u32 THREADS>
__global__ void digit_5(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ uint4 lastword[NSLOTS];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;
	__shared__ u32 next_pair;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	if (threadid < NRESTS)
		ht_len[threadid] = 0;
	else if (threadid == (THREADS - 1))
		pairs_len = 0;
	else if (threadid == (THREADS - 33))
		next_pair = 0;

    pairs_len = 0;
    next_pair = 0;

	u32 bsize = umin(eq->edata.nslots[4][bucketid], NSLOTS);

	u32 hr[2];
	int pos[2];
	pos[0] = pos[1] = SSM;

	u32 si[2];
	uint4 tt[2];
#ifdef OPT_SYNC_ALL
	__syncthreads();
#endif
#pragma unroll 2
	for (u32 i = 0; i < 2; i++)
	{
		si[i] = i * THREADS + threadid;
		if (si[i] >= bsize) break;

		slotsmall &xs = eq->round4trees[bucketid][si[i]];

		tt[i] = *(uint4*)(&xs.hash[0]);
		lastword[si[i]] = tt[i];

		asm("bfe.u32 %0, %1, 21, %2;" : "=r"(hr[i]) : "r"(tt[i].x), "r"(RB));
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	__syncthreads();
	u32 xors[4];
	u32 xorbucketid, xorslot;

#pragma unroll 2
	for (u32 i = 0; i < 2; i++)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u16 p = ht[hr[i]][0];

			*(uint4*)(&xors[0]) = tt[i] ^ lastword[p];
				
            asm("bfe.u32 %0, %1, 3, %2;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(BUCKBITS));

			xorslot = atomicAdd(&eq->edata.nslots[5][xorbucketid], 1);
			if (xorslot < NSLOTS)
			{
				slotsmall &xs = eq->round5trees[xorbucketid][xorslot];

                uint4 ttx;
                ttx.x = __byte_perm(xors[0], xors[1], 0x0765);
                ttx.y = __byte_perm(xors[1], xors[2], 0x0765);
                ttx.z = 0;
                ttx.w = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB);
                *(uint4*)(&xs.hash[0]) = ttx;
			}

			for (int k = 1; k != pos[i]; ++k)
			{
				u32 pindex = atomicAdd(&pairs_len, 1);
				if (pindex >= MAXPAIRS) break;
				u16 prev = ht[hr[i]][k];
				pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
			}
		}
	}

	__syncthreads();

	// process pairs
	u32 plen = umin(pairs_len, MAXPAIRS);
	u32 i, k;
	for (u32 s = atomicAdd(&next_pair, 1); s < plen; s = atomicAdd(&next_pair, 1))
	{
		int pair = pairs[s];
		i = __byte_perm(pair, 0, 0x4510);
		k = __byte_perm(pair, 0, 0x4532);

		*(uint4*)(&xors[0]) = lastword[i] ^ lastword[k];

        asm("bfe.u32 %0, %1, 3, %2;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(BUCKBITS));
        xorslot = atomicAdd(&eq->edata.nslots[5][xorbucketid], 1);
        if (xorslot < NSLOTS)
        {
            slotsmall &xs = eq->round5trees[xorbucketid][xorslot];

            uint4 ttx;
            ttx.x = __byte_perm(xors[0], xors[1], 0x0765);
            ttx.y = __byte_perm(xors[1], xors[2], 0x0765);
            ttx.z = 0;

            ttx.w = PACKER::set_bucketid_and_slots(bucketid, i, k, RB);
            *(uint4*)(&xs.hash[0]) = ttx;
        }
	}
}

template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS>
__global__ void digit_6(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ uint2 lastword1[NSLOTS];
	__shared__ int ht_len[MAXPAIRS];
	__shared__ u32 pairs_len;
	__shared__ u32 bsize_sh;
	__shared__ u32 next_pair;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	ht_len[threadid] = 0;
	if (threadid == (NRESTS - 1))
	{
		pairs_len = 0;
		next_pair = 0;
	}
	else if (threadid == (NRESTS - 33))
		bsize_sh = umin(eq->edata.nslots[5][bucketid], NSLOTS);

    pairs_len = 0;
    next_pair = 0;

	u32 hr[3];
	int pos[3];
	pos[0] = pos[1] = pos[2] = SSM;

	u32 si[3];
	uint2 tt[3];

	__syncthreads();

    bsize_sh = umin(eq->edata.nslots[5][bucketid], NSLOTS);
	u32 bsize = bsize_sh;

#pragma unroll 3
	for (u32 i = 0; i < 3; i++)
	{
		si[i] = i * NRESTS + threadid;
		if (si[i] >= bsize) break;

		slotsmall &xs = eq->round5trees[bucketid][si[i]];

		tt[i] = *(uint2*)(&xs.hash[0]);
		lastword1[si[i]] = tt[i];

		asm("bfe.u32 %0, %1, 22, %2;" : "=r"(hr[i]) : "r"(tt[i].x), "r"(RB));
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	// doing this to save shared memory
	int* pairs = ht_len;
	__syncthreads();

	u32 xors[3];
	u32 xorbucketid, xorslot;

#pragma unroll 3
	for (u32 i = 0; i < 3; i++)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u16 p = ht[hr[i]][0];

			*(uint2*)(&xors[0]) = *(uint2*)(&tt[i].x) ^ lastword1[p];

            asm("bfe.u32 %0, %1, 4, %2;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(BUCKBITS));
            xorslot = atomicAdd(&eq->edata.nslots[6][xorbucketid], 1);
			if (xorslot < NSLOTS)
			{
				slottiny &xst = eq->round6trees[xorbucketid][xorslot];

				uint2 ttx;
                ttx.x = __byte_perm(xors[0], xors[1], 0x0765);
				ttx.y = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB);
				*(uint2*)(&xst.hash[0]) = ttx;
			}

			if (pos[i] > 1)
			{
				p = ht[hr[i]][1];
				*(uint2*)(&xors[0]) = *(uint2*)(&tt[i].x) ^ lastword1[p];

                asm("bfe.u32 %0, %1, 4, %2;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(BUCKBITS));
				xorslot = atomicAdd(&eq->edata.nslots[6][xorbucketid], 1);
				if (xorslot < NSLOTS)
				{
                    slottiny &xst = eq->round6trees[xorbucketid][xorslot];

                    uint2 ttx;
                    ttx.x = __byte_perm(xors[0], xors[1], 0x0765);
                    ttx.y = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB);
                    *(uint2*)(&xst.hash[0]) = ttx;                    
				}

				for (int k = 2; k != pos[i]; ++k)
				{
					u32 pindex = atomicAdd(&pairs_len, 1);
					if (pindex >= MAXPAIRS) break;
					u16 prev = ht[hr[i]][k];
					pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
				}
			}
		}
	}

	__syncthreads();

	// process pairs
	u32 plen = umin(pairs_len, MAXPAIRS);
	for (u32 s = atomicAdd(&next_pair, 1); s < plen; s = atomicAdd(&next_pair, 1))
	{
		u32 pair = pairs[s];
		u32 i = __byte_perm(pair, 0, 0x4510);
		u32 k = __byte_perm(pair, 0, 0x4532);

		*(uint2*)(&xors[0]) = lastword1[i] ^ lastword1[k];

        asm("bfe.u32 %0, %1, 4, %2;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(BUCKBITS));

		xorslot = atomicAdd(&eq->edata.nslots[6][xorbucketid], 1);
		if (xorslot >= NSLOTS) continue;

        slottiny &xst = eq->round6trees[xorbucketid][xorslot];

        uint2 ttx;
        ttx.x = __byte_perm(xors[0], xors[1], 0x0765);
        ttx.y = PACKER::set_bucketid_and_slots(bucketid, i, k, RB);
        *(uint2*)(&xst.hash[0]) = ttx;
	}
}

/*
  Last round function is similar to previous ones but has different ending.
  We use warps to process final candidates. Each warp process one candidate.
  First two bidandsids (u32 of stored bucketid and two slotids) are retreived by
  lane 0 and lane 16, next four bidandsids by lane 0, 8, 16 and 24, ... until
  all lanes in warp have bidandsids from round 4. Next, each thread retreives
  16 indices. While doing so, indices are put into comparison using atomicExch
  to determine if there are duplicates (tromp's method). At the end, if no
  duplicates are found, candidate solution is saved (all indices). Note that this
  dup check method is not exact so CPU dup checking is needed after.
*/
template <u32 RB, u32 SM, int SSM, u32 FCT, typename PACKER, u32 MAXPAIRS, u32 DUPBITS, u32 W>
__global__ void digit_last_wdc(equi<RB, SM>* eq)
{
	__shared__ u8 shared_data[8192];
	int* ht_len = (int*)(&shared_data[0]);
	int* pairs = ht_len;
	u32* lastword = (u32*)(&shared_data[256 * 4]);
	u16* ht = (u16*)(&shared_data[256 * 4 + RB_NSLOTS_LD * 4]);
	u32* pairs_len = (u32*)(&shared_data[8188]);

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
#pragma unroll
	for (u32 i = 0; i < FCT; i++)
	{
		//ht_len[(i * (256 / FCT)) + threadid] = 0;
		ht_len[(i * (64 / FCT)) + threadid] = 0;
	}

	//if (threadid == ((256 / FCT) - 1))
	if (threadid == ((64 / FCT) - 1))
		*pairs_len = 0;

	slottiny* buck = eq->round6trees[bucketid];
	u32 bsize = umin(eq->edata.nslots[6][bucketid], RB_NSLOTS_LD);

	u32 si[3 * FCT];
	u32 hr[3 * FCT];
	int pos[3 * FCT];
	u32 lw[3 * FCT];

#pragma unroll
	for (u32 i = 0; i < (3 * FCT); i++)
		pos[i] = SSM;

	__syncthreads();

#pragma unroll
	for (u32 i = 0; i < (3 * FCT); i++)
	{
		//si[i] = i * (256 / FCT) + threadid;
		si[i] = i * (64 / FCT) + threadid;
		if (si[i] >= bsize) break;

		const slottiny* pslot1 = buck + si[i];

		// get xhash
		uint2 tt = *(uint2*)(&pslot1->hash[0]);
		lw[i] = tt.x;
		lastword[si[i]] = lw[i];

		u32 a;
		asm("bfe.u32 %0, %1, 23, 5;" : "=r"(a) : "r"(lw[i]));
		hr[i] = a;

		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1))
			ht[hr[i] * (SSM - 1) + pos[i]] = si[i];
	}

	__syncthreads();

#pragma unroll
	for (u32 i = 0; i < (3 * FCT); i++)
	{
		if (pos[i] >= SSM) continue;

		for (int k = 0; k != pos[i]; ++k)
		{
			u16 prev = ht[hr[i] * (SSM - 1) + k];
			if (lw[i] != lastword[prev]) continue;
			u32 pindex = atomicAdd(pairs_len, 1);
			if (pindex >= MAXPAIRS) break;
			
            pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
		}
	}


    __syncthreads();
	u32 plen = umin(*pairs_len, MAXPAIRS);

	u32 lane = threadIdx.x & 0x1f;
	u32 par = threadIdx.x >> 5;

	u32* levels = (u32*)&pairs[MAXPAIRS + (par << DUPBITS)];
	u32* susp = levels;

	while (par < plen)
	{
		int pair = pairs[par];
		par += W;

        u32 slot00 = 0; 
		if (lane % 16 == 0)
		{
			u32 plvl;
			if (lane == 0) 
                slot00 = __byte_perm(pair, 0, 0x4510);
            else
                slot00 = __byte_perm(pair, 0, 0x4532);
            
            plvl = buck[slot00].hash[1]; // slot1 r6

			slotsmall* bucks = eq->round5trees[PACKER::get_bucketid(plvl, RB, SM)];

			u32 slot1 = PACKER::get_slot1(plvl, RB, SM);
			u32 slot0 = PACKER::get_slot0(plvl, slot1, RB, SM);

			levels[lane]        = bucks[slot1].hash[3]; // slot1 r5
			levels[lane + 8]    = bucks[slot0].hash[3]; // slot0 r5
		}

        if (lane % 8 == 0)
        {
            u32 plvl = levels[lane];
            slotsmall* bucks = eq->round4trees[PACKER::get_bucketid(plvl, RB, SM)]; // r4
            u32 slot1 = PACKER::get_slot1(plvl, RB, SM);
            u32 slot0 = PACKER::get_slot0(plvl, slot1, RB, SM);

            levels[lane] = bucks[slot1].hash[3];
            levels[lane+4] = bucks[slot0].hash[3];
        }

		if (lane % 4 == 0)
        {
            u32 plvl = levels[lane];
            u32 slot1 = PACKER::get_slot1(plvl, RB, SM);
            u32 slot0 = PACKER::get_slot0(plvl, slot1, RB, SM);
			levels[lane] = eq->round3trees[PACKER::get_bucketid(plvl, RB, SM)][slot1].hash[5];
			levels[lane + 2] = eq->round3trees[PACKER::get_bucketid(plvl, RB, SM)][slot0].hash[5];
        }

		if (lane % 2 == 0)
        {
            u32 plvl = levels[lane]; 
            u32 slot1 = PACKER::get_slot1(plvl, RB, SM); 
            u32 slot0 = PACKER::get_slot0(plvl, slot1, RB, SM); 
            
			levels[lane] = eq->round2trees[PACKER::get_bucketid(plvl, RB, SM)][slot1].hash[5];
			levels[lane + 1] = eq->round2trees[PACKER::get_bucketid(plvl, RB, SM)][slot0].hash[5];
		}

		u32 ind[4];

		u32 f1 = levels[lane];
		const u32 slot1_v4 = PACKER::get_slot1(f1, RB, SM);
		const u32 slot0_v4 = PACKER::get_slot0(f1, slot1_v4, RB, SM);

		susp[lane] = 0xffffffff;
		susp[lane + 32] = 0xffffffff;
		susp[lane + 64] = 0xffffffff;
		susp[lane + 96] = 0xffffffff;
        susp[128 + lane] = 0xffffffff;
        susp[160 + lane] = 0xffffffff;
        susp[192 + lane] = 0xffffffff;
        susp[224 + lane] = 0xffffffff;

#define CHECK_DUP(a) \
	__any(atomicExch(&susp[(ind[a] & ((1 << DUPBITS) - 1))], (ind[a] >> DUPBITS)) == (ind[a] >> DUPBITS))

		u32 f2 = eq->round1trees[PACKER::get_bucketid(f1, RB, SM)][slot1_v4].hash[5];
		const slot* buck_v3_1 = &eq->round0trees[PACKER::get_bucketid(f2, RB, SM)][0]; // r0
		const u32 slot1_v3_1 = PACKER::get_slot1(f2, RB, SM);
		const u32 slot0_v3_1 = PACKER::get_slot0(f2, slot1_v3_1, RB, SM);

        ind[0] = buck_v3_1[slot1_v3_1].hash[7];
        if (CHECK_DUP(0)) continue;
        ind[1] = buck_v3_1[slot0_v3_1].hash[7];
        if (CHECK_DUP(1)) continue;

		u32 f8 = eq->round1trees[PACKER::get_bucketid(f1, RB, SM)][slot0_v4].hash[5]; // r1
        const slot* buck_v3_2 = &eq->round0trees[PACKER::get_bucketid(f8, RB, SM)][0]; // r0
        const u32 slot1_v3_2 = PACKER::get_slot1(f8, RB, SM);
        const u32 slot0_v3_2 = PACKER::get_slot0(f8, slot1_v3_2, RB, SM);

        ind[2] = buck_v3_2[slot1_v3_2].hash[7];
        if (CHECK_DUP(2)) continue;
        ind[3] = buck_v3_2[slot0_v3_2].hash[7];
        if (CHECK_DUP(3)) continue;
		u32 soli;
		if (lane == 0) 
        {
			soli = atomicAdd(&eq->srealcont.nsols, 1);
		}
#if __CUDA_ARCH__ >= 300
		// all threads get the value from lane 0
		soli = __shfl2(soli, 0);
#else
		__syncthreads();
		soli = eq->srealcont.nsols;
#endif
		if (soli < MAXREALSOLS)
		{
			if ( lane == 0)
				*(uint2*)(&eq->srealcont.sols[soli][0]) = *(uint2*)(&eq->edata.nonce[0]);

			u32 pos = 4 + (lane << 2); // (0-31) * 4
			*(uint4*)(&eq->srealcont.sols[soli][pos]) = *(uint4*)(&ind[ 0]);
		}        
	}
}

//std::mutex dev_init;
int dev_init_done[MAX_GPUS] = { 0 };

__host__
static int compu32(const void *pa, const void *pb)
{
	uint32_t a = *(uint32_t *)pa, b = *(uint32_t *)pb;
	return a<b ? -1 : a == b ? 0 : +1;
}

__host__
static bool duped(uint32_t* prf)
{
	uint32_t sortprf[128];
	memcpy(sortprf, prf, sizeof(uint32_t) * 128);
	qsort(sortprf, 128, sizeof(uint32_t), &compu32);
	for (uint32_t i = 1; i<128; i++) {
		if (sortprf[i] <= sortprf[i - 1])
        {
			return true;
        }
	}
	return false;
}

__host__
static void sort_pair(uint32_t *a, uint32_t len)
{
	uint32_t *b = a + len;
	uint32_t  tmp, need_sorting = 0;
	for (uint32_t i = 0; i < len; i++) {
		if (need_sorting || a[i] > b[i])
		{
			need_sorting = 1;
			tmp = a[i];
			a[i] = b[i];
			b[i] = tmp;
		}
		else if (a[i] < b[i])
			return;
	}
}

__host__
static void setheader(blake2b_state *ctx, const char *header, const u32 headerLen, const char* nonce, const u32 nonceLen)
{
	uint32_t le_N = WN;
	uint32_t le_K = WK;
	uchar personal[] = "FABcoin_01230123";
		    
	memcpy(personal + 8, &le_N, 4);
	memcpy(personal + 12, &le_K, 4);
	blake2b_param P[1];
	P->digest_length = HASHOUT;
	P->key_length = 0;
	P->fanout = 1;
	P->depth = 1;
	P->leaf_length = 0;
	P->node_offset = 0;
	P->node_depth = 0;
	P->inner_length = 0;
	memset(P->reserved, 0, sizeof(P->reserved));
	memset(P->salt, 0, sizeof(P->salt));
	memcpy(P->personal, (const uint8_t *)personal, 16);
	eq_blake2b_init_param(ctx, P);
	eq_blake2b_update(ctx, (const uchar *)header, headerLen);
	if (nonceLen) eq_blake2b_update(ctx, (const uchar *)nonce, nonceLen);
}

#ifdef _WIN32
dec_cuDeviceGet _cuDeviceGet = nullptr;
dec_cuCtxCreate _cuCtxCreate = nullptr;
dec_cuCtxPushCurrent _cuCtxPushCurrent = nullptr;
dec_cuCtxDestroy _cuCtxDestroy = nullptr;
#endif

template <u32 RB, u32 SM, u32 SSM, u32 THREADS, typename PACKER>
__host__ eq_cuda_context<RB, SM, SSM, THREADS, PACKER>::eq_cuda_context(int thr_id, int dev_id, fn_validate validate, fn_cancel cancel)
{   
	thread_id = thr_id;
	device_id = dev_id;
	solutions = nullptr;
	equi_mem_sz = 0;
	throughput = NBLOCKS;
	totalblocks = NBLOCKS/FD_THREADS;
	threadsperblock = FD_THREADS;
	threadsperblock_digits = THREADS;

    m_fnValidate = validate;
    m_fnCancel = cancel;

    m_buf = NULL;

	if (!dev_init_done[device_id])
	{
		// only first thread shall init device
		checkCudaErrors(hipSetDevice(device_id));
		checkCudaErrors(hipDeviceReset());
		checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

		pctx = nullptr;
	}
	else
	{
		// create new context
		hipDevice_t dev;

#ifdef _WIN32
		if (_cuDeviceGet == nullptr)
		{
			HMODULE hmod = LoadLibraryA("nvcuda.dll");
			if (hmod == NULL)
				throw std::runtime_error("Failed to load nvcuda.dll");
			_cuDeviceGet = (dec_cuDeviceGet)GetProcAddress(hmod, "hipDeviceGet");
			if (_cuDeviceGet == nullptr)
				throw std::runtime_error("Failed to get hipDeviceGet address");
			_cuCtxCreate = (dec_cuCtxCreate)GetProcAddress(hmod, "hipCtxCreate");
			if (_cuCtxCreate == nullptr)
				throw std::runtime_error("Failed to get hipCtxCreate address");
			_cuCtxPushCurrent = (dec_cuCtxPushCurrent)GetProcAddress(hmod, "hipCtxPushCurrent");
			if (_cuCtxPushCurrent == nullptr)
				throw std::runtime_error("Failed to get hipCtxPushCurrent address");
			_cuCtxDestroy = (dec_cuCtxDestroy)GetProcAddress(hmod, "hipCtxDestroy");
			if (_cuCtxDestroy == nullptr)
				throw std::runtime_error("Failed to get hipCtxDestroy address");
		}

		checkCudaDriverErrors(_cuDeviceGet(&dev, device_id));
		checkCudaDriverErrors(_cuCtxCreate(&pctx, hipDeviceScheduleBlockingSync, dev));
		checkCudaDriverErrors(_cuCtxPushCurrent(pctx));
#else
		checkCudaDriverErrors(hipDeviceGet(&dev, device_id));
		checkCudaDriverErrors(hipCtxCreate(&pctx, hipDeviceScheduleBlockingSync, dev));
		checkCudaDriverErrors(hipCtxPushCurrent(pctx));
#endif
	}
	++dev_init_done[device_id];
	//dev_init.unlock();

	checkCudaErrors(hipHostMalloc((void**)&eq, sizeof(equi<RB, SM>)));

	equi_mem_sz += sizeof(equi<RB, SM>);
	checkCudaErrors(hipMalloc((void**)&device_eq, sizeof(equi<RB, SM>)));

	equi_mem_sz += sizeof(bucket)*NBUCKETS;
	checkCudaErrors(hipMalloc((void**)&eq->round0trees, sizeof(bucket)*NBUCKETS));

	equi_mem_sz += sizeof(bucket)*NBUCKETS;
	checkCudaErrors(hipMalloc((void**)&eq->round1trees, sizeof(bucket)*NBUCKETS));

	equi_mem_sz += sizeof(bucket)*NBUCKETS;
	checkCudaErrors(hipMalloc((void**)&eq->round2trees, sizeof(bucket)*NBUCKETS));

	equi_mem_sz += sizeof(bucket)*NBUCKETS;
	checkCudaErrors(hipMalloc((void**)&eq->round3trees, sizeof(bucket)*NBUCKETS));

	equi_mem_sz += sizeof(bucketsmall)*NBUCKETS;
	checkCudaErrors(hipMalloc((void**)&eq->round4trees, sizeof(bucketsmall)*NBUCKETS));

	equi_mem_sz += sizeof(bucketsmall)*NBUCKETS;
	checkCudaErrors(hipMalloc((void**)&eq->round5trees, sizeof(bucketsmall)*NBUCKETS));

	equi_mem_sz += sizeof(buckettiny)*NBUCKETS;
	checkCudaErrors(hipMalloc((void**)&eq->round6trees, sizeof(buckettiny)*NBUCKETS));

	equi_mem_sz += 128;
	checkCudaErrors(hipMalloc((void**)&m_buf, 128));

	checkCudaErrors(hipHostMalloc((void**)&solutions, sizeof(scontainerreal)));
}

std::vector<unsigned char> GetMinimalFromIndices(std::vector<uint32_t> indices,
                                                 size_t cBitLen);

template <u32 RB, u32 SM, u32 SSM, u32 THREADS, typename PACKER>
__host__ bool eq_cuda_context<RB, SM, SSM, THREADS, PACKER>::solve(
	unsigned char *pblock,
	unsigned char *header,
	unsigned int headerlen)
{
	blake2b_state blake_ctx;                  
	int blocks = NBUCKETS;
    uchar *ptrnonce = header+headerlen-32;

	checkCudaErrors(hipMemset(device_eq, 0, sizeof(equi<RB, SM>)));
	checkCudaErrors(hipMemcpy(device_eq, eq, sizeof(equi<RB, SM>), hipMemcpyHostToDevice));

	setheader(&blake_ctx, (const char *)header, headerlen - 32, (const char *)ptrnonce, 32);
    checkCudaErrors(hipMemcpy(m_buf, blake_ctx.buf, blake_ctx.buflen, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&device_eq->blake_h, &blake_ctx.h, sizeof(u64) * 8, hipMemcpyHostToDevice));      
	{
		checkCudaErrors(hipMemset(&device_eq->edata, 0, sizeof(device_eq->edata)));		

	    digit_first<RB, SM, PACKER> <<<NBLOCKS / FD_THREADS, FD_THREADS >>>(device_eq, m_buf, *(u32 *)(ptrnonce+12));
	    if (m_fnCancel()) return false;

        digit_1<RB, SM, SSM, PACKER, NRESTS<<6, 64> <<<blocks, 64>>>(device_eq);
	    if (m_fnCancel()) return false;
		
		digit_2<RB, SM, SSM, PACKER, NRESTS << 6, 64> << <blocks, 64 >> >(device_eq);
	    if (m_fnCancel()) return false;

        digit_3<RB, SM, SSM, PACKER, NRESTS<<6, 64> <<<blocks, 64 >>>(device_eq);
	    if (m_fnCancel()) return false;

        digit_4<RB, SM, SSM, PACKER, NRESTS<<6, 64> <<<blocks, 64 >>>(device_eq);
	    if (m_fnCancel()) return false;

        digit_5<RB, SM, SSM, PACKER, NRESTS<<6, 64 > <<<blocks, 64 >>>(device_eq);
	    if (m_fnCancel()) return false;

        digit_6<RB, SM, SSM - 1, PACKER, NRESTS<<6 > <<<blocks, NRESTS >>>(device_eq);
	    if (m_fnCancel()) return false;

		digit_last_wdc<RB, SM, SSM - 3, 2, PACKER, 512, 5, 4> << <blocks, 32 >> >(device_eq);
		*(u32 *)(ptrnonce+12) = (*(u32 *)(ptrnonce+12)) + 1;
	}
    checkCudaErrors(hipMemcpy(solutions, &device_eq->srealcont, (MAXREALSOLS * (132 * 4)) + 4, hipMemcpyDeviceToHost));    	

    for (u32 s = 0; (s < solutions->nsols) && (s < MAXREALSOLS); s++)
	{
		// remove dups on CPU (dup removal on GPU is not fully exact and can pass on some invalid solutions)
		if (duped(solutions->sols[s] + 4))
		{
//			printf("duplicated\n");
			continue;
		}

		// perform sort of pairs
		for (uint32_t level = 0; level < 7; level++)
			for (uint32_t i = 0; i < (1 << 7); i += (2 << level))
				sort_pair(&solutions->sols[s][i+4], 1 << level);

        std::vector<uint32_t> index_vector(PROOFSIZE+4);		
        for (u32 i = 0; i < PROOFSIZE+4; i++) 
        {
			index_vector[i] = solutions->sols[s][i];
		}
		std::vector<unsigned char> sol_char = GetMinimalFromIndices(std::vector<uint32_t>(index_vector.begin()+4,index_vector.end()), DIGITBITS);
		if (m_fnValidate(sol_char, pblock, thread_id)) 
        {
             // If we find a POW solution, do not try other solutions
             // because they become invalid as we created a new block in blockchain.
             return true;
        }
	}
    return false;
}

// destructor
template <u32 RB, u32 SM, u32 SSM, u32 THREADS, typename PACKER>
__host__
void eq_cuda_context<RB, SM, SSM, THREADS, PACKER>::freemem()
{
	if (device_eq) {
		hipFree(device_eq);
		device_eq = NULL;
	}

    if (m_buf) {
        hipFree(m_buf);
        m_buf = NULL;
    }

	checkCudaErrors(hipFree(eq->round0trees));
	checkCudaErrors(hipFree(eq->round1trees));
	checkCudaErrors(hipFree(eq->round2trees));
	checkCudaErrors(hipFree(eq->round3trees));
	checkCudaErrors(hipFree(eq->round4trees));
	checkCudaErrors(hipFree(eq->round5trees));
	checkCudaErrors(hipFree(eq->round6trees));

	checkCudaErrors(hipHostFree(solutions));
	checkCudaErrors(hipHostFree(eq));

    if (pctx) {
		// non primary thread, destroy context
#ifdef _WIN32
		checkCudaDriverErrors(_cuCtxDestroy(pctx));
#else
		checkCudaDriverErrors(hipCtxDestroy(pctx));
#endif
	} else {
		checkCudaErrors(hipDeviceReset());
		dev_init_done[device_id] = 0;
	}
}

template <u32 RB, u32 SM, u32 SSM, u32 THREADS, typename PACKER>
__host__
eq_cuda_context<RB, SM, SSM, THREADS, PACKER>::~eq_cuda_context()
{
	freemem();
}

#ifdef CONFIG_MODE_184x7
template class eq_cuda_context<CONFIG_MODE_184x7>;
#endif
