#include "hip/hip_runtime.h"
// Equihash CUDA solver
// Copyright (c) 2016 John Tromp

#define htole32(x) (x)
#define HAVE_DECL_HTOLE32 1

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <functional>
#include <vector>
#include <iostream>

#include "eqcuda.hpp"
#include "blake2b.cu"

#define WN	184
#define WK	7
#ifndef MAX_GPUS
#define MAX_GPUS 128
#endif
#define RESTBITS	2 

#define NDIGITS		(WK+1)
#define DIGITBITS	(WN/(NDIGITS))
#define BASE        (1<<DIGITBITS)
#define NHASHES     (2*BASE)
#define HASHESPERBLAKE (512/WN)
#define HASHOUT     (HASHESPERBLAKE*WN/8)
#define BUCKBITS    (DIGITBITS-RESTBITS)

#ifndef SAVEMEM
#if RESTBITS == 4 || RESTBITS == 2
// can't save memory in such small buckets
#define SAVEMEM 1
#elif RESTBITS >= 8
// take advantage of law of large numbers (sum of 2^8 random numbers)
// this reduces (200,9) memory to under 144MB, with negligible discarding
#define SAVEMEM 9/14
#endif
#endif

// number of buckets
static const u32 NBUCKETS = 1 << BUCKBITS;
// bucket mask
// 2_log of number of slots per bucket
static const u32 SLOTBITS = RESTBITS + 1 + 1;
static const u32 SLOTRANGE = 1 << SLOTBITS;
// number of slots per bucket
static const u32 NSLOTS = SLOTRANGE * SAVEMEM;
// SLOTBITS mask
static const u32 SLOTMASK = SLOTRANGE - 1;
// number of possible values of xhash (rest of n) bits
static const u32 NRESTS = 1 << RESTBITS;
// RESTBITS mask
static const u32 RESTMASK = NRESTS - 1;
// number of blocks of hashes extracted from single 512 bit blake2b output
static const u32 NBLOCKS = (NHASHES + HASHESPERBLAKE - 1) / HASHESPERBLAKE;
// nothing larger found in 100000 runs
static const u32 MAXSOLS = 10;

static const u32 HASHLEN = (WN+7)/8;

//std::mutex dev_init;
int dev_init_done184[MAX_GPUS] = { 0 };

void setheader(blake2b_state *ctx, const unsigned char *header, const u32 headerLen, const unsigned char* nce, const u32 nonceLen) 
{
  uint32_t le_N = WN;
  uint32_t le_K = WK;
  uchar personal[] = "ZcashPoW01230123";
  
  if( le_N != 200 || le_K != 9 )
      memcpy(personal, "FABcoin_", 8);

  memcpy(personal+8,  &le_N, 4);
  memcpy(personal+12, &le_K, 4);
  blake2b_param P[1];
  P->digest_length = HASHOUT;
  P->key_length    = 0;
  P->fanout        = 1;
  P->depth         = 1;
  P->leaf_length   = 0;
  P->node_offset   = 0;
  P->node_depth    = 0;
  P->inner_length  = 0;
  memset(P->reserved, 0, sizeof(P->reserved));
  memset(P->salt,     0, sizeof(P->salt));
  memcpy(P->personal, (const uint8_t *)personal, 16);
  eq_blake2b_init_param(ctx, P);
  eq_blake2b_update(ctx, (const uchar *)header, headerLen);
  eq_blake2b_update(ctx, (const uchar *)nce, nonceLen);
}




// tree node identifying its children as two different slots in
// a bucket on previous layer with the same rest bits (x-tra hash)
struct tree {
	u32 bid_s0_s1_x; // manual bitfields

	__device__ tree(const u32 idx, const u32 xh) {
		bid_s0_s1_x = idx << RESTBITS | xh;
	}
	__device__ tree(const u32 idx) {
		bid_s0_s1_x = idx;
	}

#ifdef XINTREE
  	__device__ tree(const u32 bid, const u32 s0, const u32 s1, const u32 xh) {
  	bid_s0_s1_x = ((((bid << SLOTBITS) | s0) << SLOTBITS) | s1) << RESTBITS | xh;
#else
  	__device__ tree(const u32 bid, const u32 s0, const u32 s1) {
  	bid_s0_s1_x = (((bid << SLOTBITS) | s0) << SLOTBITS) | s1;
#endif
  }
  __device__ u32 getindex() const {
#ifdef XINTREE
		return bid_s0_s1_x >> RESTBITS;
#else
		return bid_s0_s1_x;
#endif
	}
	__device__ u32 bucketid() const {
#ifdef XINTREE
		return bid_s0_s1_x >> (2 * SLOTBITS + RESTBITS);
#else
		return bid_s0_s1_x >> (2 * SLOTBITS);
#endif
	}
	__device__ u32 slotid0() const {
#ifdef XINTREE
		return (bid_s0_s1_x >> SLOTBITS + RESTBITS) & SLOTMASK;
#else
		return (bid_s0_s1_x >> SLOTBITS) & SLOTMASK;
#endif
	}
	__device__ u32 slotid1() const {
#ifdef XINTREE
		return (bid_s0_s1_x >> RESTBITS) & SLOTMASK;
#else
		return bid_s0_s1_x & SLOTMASK;
#endif
	}
	__device__ u32 xhash() const {
		return bid_s0_s1_x & RESTMASK;
	}
  __device__ bool prob_disjoint(const tree other) const {
    tree xort(bid_s0_s1_x ^ other.bid_s0_s1_x);
    return xort.bucketid() || (xort.slotid0() && xort.slotid1());
    // next two tests catch much fewer cases and are therefore skipped
    // && slotid0() != other.slotid1() && slotid1() != other.slotid0()
  }
};

union hashunit {
	u32 word;
	uchar bytes[sizeof(u32)];
};

#define WORDS(bits)     ((bits + 31) / 32)
#define HASHWORDS0 WORDS(WN - DIGITBITS + RESTBITS)
#define HASHWORDS1 WORDS(WN - 2*DIGITBITS + RESTBITS)

struct slot0 {
	tree attr;
	hashunit hash[HASHWORDS0];
};

struct slot1 {
	tree attr;
	hashunit hash[HASHWORDS1]; 
};

// a bucket is NSLOTS treenodes
typedef slot0 bucket0[NSLOTS];
typedef slot1 bucket1[NSLOTS];
// the N-bit hash consists of K+1 n-bit "digits"
// each of which corresponds to a layer of NBUCKETS buckets
typedef bucket0 digit0[NBUCKETS];
typedef bucket1 digit1[NBUCKETS];

// size (in bytes) of hash in round 0 <= r < WK
u32 hhashsize(const u32 r) {
#ifdef XINTREE
  const u32 hashbits = WN - (r+1) * DIGITBITS;
#else
  const u32 hashbits = WN - (r+1) * DIGITBITS + RESTBITS;
#endif
	return (hashbits + 7) / 8;
}
// size (in bytes) of hash in round 0 <= r < WK
__device__ u32 hashsize(const u32 r) {
#ifdef XINTREE
  const u32 hashbits = WN - (r+1) * DIGITBITS;
  return (hashbits + 7) / 8;
#else
    const u32 hashbits = WN - (r+1) * DIGITBITS + RESTBITS;
    return (hashbits + 7) / 8;
#endif
}

u32 hhashwords(u32 bytes) {
	return (bytes + 3) / 4;
}

__device__ u32 hashwords(u32 bytes) {
	return (bytes + 3) / 4;
}

// manages hash and tree data
struct htalloc {
	bucket0 *trees0[(WK + 1) / 2];
	bucket1 *trees1[WK / 2];
};

typedef u32 bsizes[NBUCKETS];

struct equi1847 {
	blake2b_state blake_ctx;
	htalloc hta;
	bsizes *nslots;
	proof *sols;
	u32 nsols;
	u32 nthreads;
	equi1847(const u32 n_threads) {
		nthreads = n_threads;
	}
	void setheadernonce(unsigned char *header, const u32 len, unsigned char* nonce, const u32 nlen) {
		setheader(&blake_ctx, header, len, nonce, nlen);
		checkCudaErrors(hipMemset(nslots, 0, NBUCKETS * sizeof(u32)));
		nsols = 0;
	}
	__device__ u32 getnslots0(const u32 bid) {
		u32 &nslot = nslots[0][bid];
		const u32 n = min(nslot, NSLOTS);
		nslot = 0;
		return n;
	}
	__device__ u32 getnslots1(const u32 bid) {
		u32 &nslot = nslots[1][bid];
		const u32 n = min(nslot, NSLOTS);
		nslot = 0;
		return n;
	}
  __device__ bool orderindices(u32 *indices, u32 size) {
		if (indices[0] > indices[size]) {
			for (u32 i = 0; i < size; i++) {
				const u32 tmp = indices[i];
				indices[i] = indices[size + i];
				indices[size + i] = tmp;
			}
		}
    return false;
	}
  __device__ bool listindices1(const tree t, u32 *indices) {
		const bucket0 &buck = hta.trees0[0][t.bucketid()];
		const u32 size = 1 << 0;
		indices[0] = buck[t.slotid0()].attr.getindex();
		indices[size] = buck[t.slotid1()].attr.getindex();
		orderindices(indices, size);
    return false;
	}
  __device__ bool listindices2(const tree t, u32 *indices) {
    const bucket1 &buck = hta.trees1[0][t.bucketid()];
    const u32 size = 1 << 1;
    return listindices1(buck[t.slotid0()].attr, indices) || 
           listindices1(buck[t.slotid1()].attr, indices+size) ||
           orderindices(indices, size) || indices[0] == indices[size];
  }
  __device__ bool listindices3(const tree t, u32 *indices) {
    const bucket0 &buck = hta.trees0[1][t.bucketid()];
    const u32 size = 1 << 2;
    return listindices2(buck[t.slotid0()].attr, indices) || 
           listindices2(buck[t.slotid1()].attr, indices+size) ||
           orderindices(indices, size) || indices[0] == indices[size];
  }
  __device__ bool listindices4(const tree t, u32 *indices) {
    const bucket1 &buck = hta.trees1[1][t.bucketid()];
    const u32 size = 1 << 3;
    return listindices3(buck[t.slotid0()].attr, indices) || 
           listindices3(buck[t.slotid1()].attr, indices+size) ||
           orderindices(indices, size) || indices[0] == indices[size];
  }
  __device__ bool listindices5(const tree t, u32 *indices) {
    const bucket0 &buck = hta.trees0[2][t.bucketid()];
    const u32 size = 1 << 4;
    return listindices4(buck[t.slotid0()].attr, indices) || 
           listindices4(buck[t.slotid1()].attr, indices+size) ||
           orderindices(indices, size) || indices[0] == indices[size];
  }

  __device__ bool listindices6(const tree t, u32 *indices) {
    const bucket1 &buck = hta.trees1[2][t.bucketid()];
    const u32 size = 1 << 5;
    return listindices5(buck[t.slotid0()].attr, indices) || 
           listindices5(buck[t.slotid1()].attr, indices+size) ||
           orderindices(indices, size) || indices[0] == indices[size];
  }
  __device__ bool listindices7(const tree t, u32 *indices) {
    const bucket0 &buck = hta.trees0[3][t.bucketid()];
    const u32 size = 1 << 6;
    return listindices6(buck[t.slotid0()].attr, indices) || 
           listindices6(buck[t.slotid1()].attr, indices+size) ||
           orderindices(indices, size) || indices[0] == indices[size];
  }

#if WK >= 8
  __device__ bool listindices8(const tree t, u32 *indices) {
    const bucket1 &buck = hta.trees1[3][t.bucketid()];
    const u32 size = 1 << 7;
    return listindices7(buck[t.slotid0()].attr, indices) || 
           listindices7(buck[t.slotid1()].attr, indices+size) ||
           orderindices(indices, size) || indices[0] == indices[size];
  }
  __device__ bool listindices9(const tree t, u32 *indices) {
    const bucket0 &buck = hta.trees0[4][t.bucketid()];
    const u32 size = 1 << 8;
    return listindices8(buck[t.slotid0()].attr, indices) || 
           listindices8(buck[t.slotid1()].attr, indices+size) ||
           orderindices(indices, size) || indices[0] == indices[size];
  }
#endif

  __device__ void candidate(const tree t) {
    proof prf;
#if WK==9
    if (listindices9(t, prf)) return;
#elif WK==7
    if (listindices7(t, prf)) return;
#elif WK==5
    if (listindices5(t, prf)) return;
#else
#error not implemented
#endif
    u32 soli = atomicAdd(&nsols, 1);
    if (soli < MAXSOLS)
#if WK==9
      listindices9(t, sols[soli]);
#elif WK==7
      listindices7(t, sols[soli]);
#elif WK==5
      listindices5(t, sols[soli]);
#else
#error not implemented
#endif
	}
	void showbsizes(u32 r) {
#if defined(HIST) || defined(SPARK) || defined(LOGSPARK)
		u32 ns[NBUCKETS];
		checkCudaErrors(hipMemcpy(ns, nslots[r & 1], NBUCKETS * sizeof(u32), hipMemcpyDeviceToHost));
		u32 binsizes[65];
		memset(binsizes, 0, 65 * sizeof(u32));
		for (u32 bucketid = 0; bucketid < NBUCKETS; bucketid++) {
			u32 bsize = min(ns[bucketid], NSLOTS) >> (SLOTBITS - 6);
			binsizes[bsize]++;
		}
		for (u32 i = 0; i < 65; i++) {
#ifdef HIST
			printf(" %d:%d", i, binsizes[i]);
#else
#ifdef SPARK
			u32 sparks = binsizes[i] / SPARKSCALE;
#else
			u32 sparks = 0;
			for (u32 bs = binsizes[i]; bs; bs >>= 1) sparks++;
			sparks = sparks * 7 / SPARKSCALE;
#endif
			printf("\342\226%c", '\201' + sparks);
#endif
		}
		printf("\n");
#endif
  }
  struct htlayout {
    htalloc hta;
    u32 prevhashunits;
    u32 nexthashunits;
    u32 dunits;
    u32 prevbo;
    u32 nextbo;

		__device__ htlayout(equi1847 *eq, u32 r) : hta(eq->hta), prevhashunits(0), dunits(0) {
			u32 nexthashbytes = hashsize(r);
			nexthashunits = hashwords(nexthashbytes);
			prevbo = 0;
			nextbo = nexthashunits * sizeof(hashunit) - nexthashbytes; // 0-3
			if (r) {
				u32 prevhashbytes = hashsize(r-1);
				prevhashunits = hashwords(prevhashbytes);
				prevbo = prevhashunits * sizeof(hashunit) - prevhashbytes; // 0-3
				dunits = prevhashunits - nexthashunits;
			}
		}
    __device__ u32 getxhash0(const slot0* pslot, int r = 0 ) const {
#ifdef XINTREE
      return pslot->attr.xhash();
#elif DIGITBITS % 8 == 4 && RESTBITS == 4
      return pslot->hash->bytes[prevbo] >> 4;
#elif DIGITBITS % 8 == 4 && RESTBITS == 6
      return (pslot->hash->bytes[prevbo] & 0x3) << 4 | pslot->hash->bytes[prevbo+1] >> 4;
#elif DIGITBITS % 8 == 4 && RESTBITS == 8
      return (pslot->hash->bytes[prevbo] & 0xf) << 4 | pslot->hash->bytes[prevbo+1] >> 4;
#elif DIGITBITS % 8 == 4 && RESTBITS == 10
      return (pslot->hash->bytes[prevbo] & 0x3f) << 4 | pslot->hash->bytes[prevbo+1] >> 4;
#elif DIGITBITS % 8 == 7 && RESTBITS == 4  // 184,7
      if( r != 0 )
      {
          switch(r)
          {
          case 1:
              return (pslot->hash->bytes[prevbo] & 0x1f) >>1;
          case 3:
              return ((pslot->hash->bytes[prevbo] & 0x7f) >>3);
          case 5:
              return ((pslot->hash->bytes[prevbo] & 0x1) << 3) | (pslot->hash->bytes[prevbo+1] >> 5);
          case 7:
              return ((pslot->hash->bytes[prevbo] & 0x7) << 1) | (pslot->hash->bytes[prevbo+1] >> 7);
          default:
              return 0;
          }
      }      
#elif DIGITBITS % 8 == 7 && RESTBITS == 2  // 184,7
        if( r != 0 )
        {
            switch(r)
            {
            case 1:
                return (pslot->hash->bytes[prevbo] & 0x07) >> 1;
            case 3:
                return ((pslot->hash->bytes[prevbo] & 0x1f) >> 3);
            case 5:
                return ((pslot->hash->bytes[prevbo] & 0x7f) >> 5);
            case 7:
                return ((pslot->hash->bytes[prevbo] & 0x1) << 1) | (pslot->hash->bytes[prevbo+1] >> 7);
            default:
                return 0;
            }
        }      
#elif DIGITBITS % 8 == 0 && RESTBITS == 4
      return pslot->hash->bytes[prevbo] & 0xf;
#elif RESTBITS == 0
      return 0;
#else
#error non implemented
#endif
        return 0;
    }
    __device__ u32 getxhash1(const slot1* pslot, int r = 0 ) const {
#ifdef XINTREE
      return pslot->attr.xhash();
#elif DIGITBITS % 4 == 0 && RESTBITS == 4
      return pslot->hash->bytes[prevbo] & 0xf;
#elif DIGITBITS % 4 == 0 && RESTBITS == 6
      return pslot->hash->bytes[prevbo] & 0x3f;
#elif DIGITBITS % 4 == 0 && RESTBITS == 8
      return pslot->hash->bytes[prevbo];
#elif DIGITBITS % 4 == 0 && RESTBITS == 10
      return (pslot->hash->bytes[prevbo] & 0x3) << 8 | pslot->hash->bytes[prevbo+1];
#elif DIGITBITS % 8 == 7 && RESTBITS == 4  // 184,7
        if( r != 0 )
        {
            switch(r)
            {
            case 2:
                return ((pslot->hash->bytes[prevbo] & 0x3f) >> 2);
            case 4:
                return pslot->hash->bytes[prevbo] >> 4;
            case 6:
                return ((pslot->hash->bytes[prevbo] & 0x3 ) << 2) | (pslot->hash->bytes[prevbo+1] >> 6);
            default:
                return 0;
            }
        }      
#elif DIGITBITS % 8 == 7 && RESTBITS == 2  // 184,7
        if( r != 0 )
        {
            switch(r)
            {
            case 2:
                return ((pslot->hash->bytes[prevbo] & 0x0f) >> 2);
            case 4:
                return ((pslot->hash->bytes[prevbo] & 0x3f) >> 4);
            case 6:
                return pslot->hash->bytes[prevbo] >> 6;
            default:
                return 0;
            }
        }      
#elif RESTBITS == 0
      return 0;
#else
#error non implemented
#endif
        return 0;
    }
    __device__ void getxorbucket(const uchar *bytes0, const uchar *bytes1, u32 &xorbucketid, u32 &xhash, int r = 0 ) const {
    
#if WN == 184 && WK == 7 && RESTBITS == 4
        switch( r )
        {
            case 1:
                xorbucketid = ((u32)(bytes0[prevbo] ^ bytes1[prevbo]) & 0x1) << 18
                            | ((u32)(bytes0[prevbo+1] ^ bytes1[prevbo+1])) << 10
                            | ((u32)(bytes0[prevbo+2] ^ bytes1[prevbo+2])) << 2
                            | (bytes0[prevbo+3] ^ bytes1[prevbo+3]) >> 6;
                break;
            case 2:
                xorbucketid = ((u32)(bytes0[prevbo] ^ bytes1[prevbo]) & 0x3) << 17
                    | ((u32)(bytes0[prevbo+1] ^ bytes1[prevbo+1])) << 9
                    | ((u32)(bytes0[prevbo+2] ^ bytes1[prevbo+2])) << 1
                    | (bytes0[prevbo+3] ^ bytes1[prevbo+3]) >> 7;
                break;
            case 3:
                xorbucketid = (((u32)(bytes0[prevbo] ^ bytes1[prevbo]) & 0x7) << 16)
                    | ((u32)(bytes0[prevbo+1] ^ bytes1[prevbo+1])) << 8
                    | (bytes0[prevbo+2] ^ bytes1[prevbo+2]);
                break;
            case 4:
                xorbucketid = ((u32)(bytes0[prevbo] ^ bytes1[prevbo]) & 0xf) << 15
                    | ((u32)(bytes0[prevbo+1] ^ bytes1[prevbo+1])) << 7
                    | (bytes0[prevbo+2] ^ bytes1[prevbo+2]) >> 1;
                break;
            case 5:
                xorbucketid = ((u32)(bytes0[prevbo+1] ^ bytes1[prevbo+1]) & 0x1f ) << 14
                    | ((u32)(bytes0[prevbo+2] ^ bytes1[prevbo+2])) << 6
                    | (bytes0[prevbo+3] ^ bytes1[prevbo+3]) >> 2;
                break;
            case 6:
                xorbucketid = (((u32)(bytes0[prevbo+1] ^ bytes1[prevbo+1]) & 0x3f ) << 13 )
                    | ((u32)(bytes0[prevbo+2] ^ bytes1[prevbo+2])) << 5
                    | (bytes0[prevbo+3] ^ bytes1[prevbo+3]) >> 3;
                break;
        }
#elif WN == 184 && WK == 7 && RESTBITS == 2
        switch( r )
        {
        case 1:
            xorbucketid = ((u32)(bytes0[prevbo] ^ bytes1[prevbo]) & 0x1) << 20
                | ((u32)(bytes0[prevbo+1] ^ bytes1[prevbo+1])) << 12
                | ((u32)(bytes0[prevbo+2] ^ bytes1[prevbo+2])) << 4
                | (bytes0[prevbo+3] ^ bytes1[prevbo+3]) >> 4;
            break;
        case 2:
            xorbucketid = ((u32)(bytes0[prevbo] ^ bytes1[prevbo]) & 0x3) << 19
                | ((u32)(bytes0[prevbo+1] ^ bytes1[prevbo+1])) << 11
                | ((u32)(bytes0[prevbo+2] ^ bytes1[prevbo+2])) << 3
                | (bytes0[prevbo+3] ^ bytes1[prevbo+3]) >> 5;
            break;
        case 3:
            xorbucketid = ((u32)(bytes0[prevbo] ^ bytes1[prevbo]) & 0x7) << 18
                | ((u32)(bytes0[prevbo+1] ^ bytes1[prevbo+1])) << 10
                | ((u32)(bytes0[prevbo+2] ^ bytes1[prevbo+2])) << 2
                | (bytes0[prevbo+3] ^ bytes1[prevbo+3]) >> 6;
            break;
        case 4:
            xorbucketid = ((u32)(bytes0[prevbo] ^ bytes1[prevbo]) & 0xf) << 17
                | ((u32)(bytes0[prevbo+1] ^ bytes1[prevbo+1])) << 9
                | ((u32)(bytes0[prevbo+2] ^ bytes1[prevbo+2])) << 1
                | (bytes0[prevbo+3] ^ bytes1[prevbo+3]) >> 7;
            break;
        case 5:
            xorbucketid = ((u32)(bytes0[prevbo+0] ^ bytes1[prevbo+0]) & 0x1f ) << 16
                | ((u32)(bytes0[prevbo+1] ^ bytes1[prevbo+1])) << 8
                | (bytes0[prevbo+2] ^ bytes1[prevbo+2]);
            break;
        case 6:
            xorbucketid = (((u32)(bytes0[prevbo+0] ^ bytes1[prevbo+0]) & 0x3f ) << 15 )
                | ((u32)(bytes0[prevbo+1] ^ bytes1[prevbo+1])) << 7
                | (bytes0[prevbo+2] ^ bytes1[prevbo+2]) >> 1;
            break;
        }
#endif
    }        

    __device__ bool equal(const hashunit *hash0, const hashunit *hash1) const {
      return hash0[prevhashunits-1].word == hash1[prevhashunits-1].word;
    }
  };

	struct collisiondata {
#ifdef XBITMAP
#if NSLOTS > 64
#error cant use XBITMAP with more than 64 slots
#endif
		u64 xhashmap[NRESTS];
		u64 xmap;
#else
#if RESTBITS <= 6
		typedef uchar xslot;
#else
		typedef u16 xslot;
#endif
		static const xslot xnil = ~0;
		xslot xhashslots[NRESTS];
		xslot nextxhashslot[NSLOTS];
		xslot nextslot;
#endif
		u32 s0;

		__device__ void clear() {
#ifdef XBITMAP
			memset(xhashmap, 0, NRESTS * sizeof(u64));
#else
			memset(xhashslots, xnil, NRESTS * sizeof(xslot));
			memset(nextxhashslot, xnil, NSLOTS * sizeof(xslot));
#endif
    }
    __device__ void addslot(u32 s1, u32 xh) {
#ifdef XBITMAP
      xmap = xhashmap[xh];
      xhashmap[xh] |= (u64)1 << s1;
      s0 = ~0;
#else
      nextslot = xhashslots[xh];
      nextxhashslot[s1] = nextslot;
      xhashslots[xh] = s1;
#endif
		}
		__device__ bool nextcollision() const {
#ifdef XBITMAP
			return xmap != 0;
#else
			return nextslot != xnil;
#endif
		}
		__device__ u32 slot() {
#ifdef XBITMAP
			const u32 ffs = __ffsll(xmap);
			s0 += ffs; xmap >>= ffs;
#else
			nextslot = nextxhashslot[s0 = nextslot];
#endif
			return s0;
		}
	};
		};

__global__ void digitH(equi1847 *eq) {
	uchar hash[HASHOUT];
	blake2b_state state;
	equi1847::htlayout htl(eq, 0);
	const u32 hashbytes = hashsize(0); // always 23 ?
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 block = id; block < NBLOCKS; block += eq->nthreads) {
		state = eq->blake_ctx;
		blake2b_gpu_hash(&state, block, hash, HASHOUT);
		for (u32 i = 0; i<HASHESPERBLAKE; i++) {
      const uchar *ph = hash + i * (HASHLEN);
#if BUCKBITS == 16 && RESTBITS == 4
			const u32 bucketid = ((u32)ph[0] << 8) | ph[1];
#ifdef XINTREE
			const u32 xhash = ph[2] >> 4;
#endif
#elif BUCKBITS == 14 && RESTBITS == 6
			const u32 bucketid = ((u32)ph[0] << 6) | ph[1] >> 2;
#elif BUCKBITS == 12 && RESTBITS == 8
      const u32 bucketid = ((u32)ph[0] << 4) | ph[1] >> 4;
#elif BUCKBITS == 19 && RESTBITS == 4 // 184,7     
      const u32 bucketid = ((((u32)ph[0] << 8) | ph[1]) << 3) | ph[2] >> 5;
#elif BUCKBITS == 21 && RESTBITS == 2 // 184,7     
      const u32 bucketid = ((((u32)ph[0] << 8) | ph[1]) << 5) | ph[2] >> 3;
#elif BUCKBITS == 20 && RESTBITS == 4
			const u32 bucketid = ((((u32)ph[0] << 8) | ph[1]) << 4) | ph[2] >> 4;
#ifdef XINTREE
			const u32 xhash = ph[2] & 0xf;
#endif
#elif BUCKBITS == 12 && RESTBITS == 4
      const u32 bucketid = ((u32)ph[0] << 4) | ph[1] >> 4;
#ifdef XINTREE
      const u32 xhash = ph[1] & 0xf;
#endif
#else
#error not implemented
#endif
			const u32 slot = atomicAdd(&eq->nslots[0][bucketid], 1);
			if (slot >= NSLOTS)
				continue;
			slot0 &s = eq->hta.trees0[0][bucketid][slot];
#ifdef XINTREE
			s.attr = tree(block*HASHESPERBLAKE+i, xhash);
#else
			s.attr = tree(block*HASHESPERBLAKE+i);
#endif
			memcpy(s.hash->bytes+htl.nextbo, ph+HASHLEN-hashbytes, hashbytes);
		}
	}
}

__global__ void digitO(equi1847 *eq, const u32 r) {
	equi1847::htlayout htl(eq, r);
	equi1847::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot0 *buck = htl.hta.trees0[(r - 1) / 2][bucketid];
		u32 bsize = eq->getnslots0(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot0 *pslot1 = buck + s1;
            u32 xhash0 = htl.getxhash0( pslot1, r );
            for (cd.addslot(s1, xhash0); cd.nextcollision(); ) 
            {
				const u32 s0 = cd.slot();
				const slot0 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				u32 xorbucketid;
				u32 xhash;
				const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;
#if WN == 200 && BUCKBITS == 16 && RESTBITS == 4 && defined(XINTREE)
				xorbucketid = ((((u32)(bytes0[htl.prevbo] ^ bytes1[htl.prevbo]) & 0xf) << 8)
					| (bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1])) << 4
					| (xhash = bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 4;
				xhash &= 0xf;
#elif WN % 24 == 0 && BUCKBITS == 20 && RESTBITS == 4
				xorbucketid = ((((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) << 8)
					| (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2])) << 4)
					| (xhash = bytes0[htl.prevbo + 3] ^ bytes1[htl.prevbo + 3]) >> 4;
				xhash &= 0xf;
#elif WN == 96 && BUCKBITS == 12 && RESTBITS == 4
				xorbucketid = ((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) << 4)
					| (xhash = bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 4;
				xhash &= 0xf;
#elif WN == 200 && BUCKBITS == 14 && RESTBITS == 6
				xorbucketid = ((((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) & 0xf) << 8)
					| (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2])) << 2
					| (bytes0[htl.prevbo + 3] ^ bytes1[htl.prevbo + 3]) >> 6;
#elif WN == 184 && WK == 7 // 184,7
                htl.getxorbucket(bytes0, bytes1, xorbucketid, xhash, r );
#else
#error not implemented
#endif
				const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot1 &xs = htl.hta.trees1[r/2][xorbucketid][xorslot];
#ifdef XINTREE
				xs.attr = tree(bucketid, s0, s1, xhash);
#else
				xs.attr = tree(bucketid, s0, s1);
#endif
				for (u32 i=htl.dunits; i < htl.prevhashunits; i++)
					xs.hash[i - htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
			}
		}
	}
}

__global__ void digitE(equi1847 *eq, const u32 r) {
	equi1847::htlayout htl(eq, r);
	equi1847::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot1 *buck = htl.hta.trees1[(r - 1) / 2][bucketid];
		u32 bsize = eq->getnslots1(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot1 *pslot1 = buck + s1;
      u32 xhash1 = htl.getxhash1( pslot1, r );
      for (cd.addslot(s1, xhash1); cd.nextcollision(); ) {
				const u32 s0 = cd.slot();
				const slot1 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				u32 xorbucketid;
				u32 xhash;
				const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;
#if WN == 200 && BUCKBITS == 16 && RESTBITS == 4 && defined(XINTREE)
        xorbucketid = ((u32)(bytes0[htl.prevbo] ^ bytes1[htl.prevbo]) << 8)
                        | (bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]);
        u32 xhash = (bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2]) >> 4;
#elif WN % 24 == 0 && BUCKBITS == 20 && RESTBITS == 4
        xorbucketid = ((((u32)(bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]) << 8)
                            | (bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2])) << 4)
                            | (bytes0[htl.prevbo+3] ^ bytes1[htl.prevbo+3]) >> 4;
#elif WN == 96 && BUCKBITS == 12 && RESTBITS == 4
				xorbucketid = ((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) << 4)
					| (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 4;
#elif WN == 200 && BUCKBITS == 14 && RESTBITS == 6
        xorbucketid = ((u32)(bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]) << 6)
                          | (bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2]) >> 2;
#elif WN == 184 && WK == 7
        htl.getxorbucket(bytes0, bytes1, xorbucketid, xhash, r );
#else
#error not implemented
#endif
				const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot0 &xs = htl.hta.trees0[r / 2][xorbucketid][xorslot];
#ifdef XINTREE
				xs.attr = tree(bucketid, s0, s1, xhash);
#else
				xs.attr = tree(bucketid, s0, s1);
#endif
				for (u32 i = htl.dunits; i < htl.prevhashunits; i++)
					xs.hash[i - htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
			}
		}
	}
}

__global__ void digitK(equi1847 *eq) {
  equi1847::collisiondata cd;
  equi1847::htlayout htl(eq, WK);
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[(WK-1)/2][bucketid];
    u32 bsize = eq->getnslots0(bucketid); // assume WK odd
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;

      u32 xhash0 = htl.getxhash0( pslot1, WK );
      
      for (cd.addslot(s1, xhash0); cd.nextcollision(); ) { // assume WK odd
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash) && pslot0->attr.prob_disjoint(pslot1->attr)) {
#ifdef XINTREE
					eq->candidate(tree(bucketid, s0, s1, 0));
#else
					eq->candidate(tree(bucketid, s0, s1));
#endif
				}
			}
		}
	}
}


eq_cuda_context1847::eq_cuda_context1847(int thrid, int devid, fn_validate validate, fn_cancel cancel)	
{
    threadsperblock = 64;
    device_id = devid;
    thread_id = thrid;

    m_fnValidate = validate;
    m_fnCancel = cancel;

    hipDeviceProp_t device_props;
    checkCudaErrors(hipGetDeviceProperties(&device_props, device_id));
    totalblocks = device_props.multiProcessorCount * 7;
    
	eq = new equi1847(threadsperblock * totalblocks);
	sol_memory = malloc(sizeof(proof) * MAXSOLS + 4096);
	solutions = (proof*)(((long long)sol_memory + 4095) & -4096);

	if (!dev_init_done184[device_id])
	{	
		checkCudaErrors(hipSetDevice(device_id));
		checkCudaErrors(hipDeviceReset());
		checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

		pctx = nullptr;
	}
	else
	{
		// create new context
		hipDevice_t dev;

		checkCudaDriverErrors(hipDeviceGet(&dev, device_id));
		checkCudaDriverErrors(hipCtxCreate(&pctx, hipDeviceScheduleBlockingSync, dev));
		checkCudaDriverErrors(hipCtxPushCurrent(pctx));
	}
	++dev_init_done184[device_id];
	
	checkCudaErrors(hipMalloc((void**)&heap0, sizeof(digit0)));
	checkCudaErrors(hipMalloc((void**)&heap1, sizeof(digit1)));

	for (u32 r = 0; r < WK; r++)
		if ((r & 1) == 0)
			eq->hta.trees0[r / 2] = (bucket0 *)(heap0 + r / 2);
		else
			eq->hta.trees1[r / 2] = (bucket1 *)(heap1 + r / 2);

	checkCudaErrors(hipMalloc((void**)&eq->nslots, 2 * NBUCKETS * sizeof(u32)));
	checkCudaErrors(hipMalloc((void**)&eq->sols, MAXSOLS * sizeof(proof)));

	checkCudaErrors(hipMalloc((void**)&device_eq, sizeof(equi1847)));
}


eq_cuda_context1847::~eq_cuda_context1847()
{
	checkCudaErrors(hipFree(eq->nslots));
	checkCudaErrors(hipFree(eq->sols));
	checkCudaErrors(hipFree(heap0));
	checkCudaErrors(hipFree(heap1));
	free(sol_memory);
	delete eq;

	if (pctx) 
	{
		// non primary thread, destroy context
#ifdef WIN32
		checkCudaDriverErrors(_cuCtxDestroy(pctx));
#else
		checkCudaDriverErrors(hipCtxDestroy(pctx));
#endif
	}
	else
	{
		checkCudaErrors(hipDeviceReset());
		dev_init_done184[device_id] = 0;
	}
}

std::vector<unsigned char> GetMinimalFromIndices(std::vector<uint32_t> indices,
                                                 size_t cBitLen);

bool eq_cuda_context1847::solve(unsigned char *pblock, unsigned char *header, unsigned int headerlen)
{
	eq->setheadernonce(header, headerlen-32, header+headerlen-32, 32);
	checkCudaErrors(hipMemcpy(device_eq, eq, sizeof(equi1847), hipMemcpyHostToDevice));

	digitH << <totalblocks, threadsperblock >> >(device_eq);
	if (m_fnCancel()) return false;

    for (u32 r = 1; r < WK; r++) {
		r & 1 ? digitO << <totalblocks, threadsperblock >> >(device_eq, r)
			: digitE << <totalblocks, threadsperblock >> >(device_eq, r);
	}

	if (m_fnCancel()) return false;
	digitK << <totalblocks, threadsperblock >> >(device_eq);

	checkCudaErrors(hipMemcpy(eq, device_eq, sizeof(equi1847), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(solutions, eq->sols, MAXSOLS * sizeof(proof), hipMemcpyDeviceToHost));

	for (unsigned s = 0; (s < eq->nsols) && (s < MAXSOLS); s++)
	{
		std::vector<uint32_t> index_vector(PROOFSIZE1);
		for (u32 i = 0; i < PROOFSIZE1; i++) {
			index_vector[i] = solutions[s][i];
		}

		std::vector<unsigned char> sol_char = GetMinimalFromIndices(index_vector, DIGITBITS);
		if (m_fnValidate(sol_char, pblock, thread_id)) 
        {
             // If we find a POW solution, do not try other solutions
             // because they become invalid as we created a new block in blockchain.
             return true;
        }        
	}
	return false;
}