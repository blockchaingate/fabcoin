#include "hip/hip_runtime.h"
// Equihash CUDA solver
// Copyright (c) 2016 John Tromp

#define htole32(x) (x)
#define HAVE_DECL_HTOLE32 1

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <functional>
#include <vector>
#include <iostream>

#include "eqcuda.hpp"
#include "blake2b.cu"

#define WN	210
#define WK	9

#if WN == 200
#define XINTREE
#define UNROLL
#define RESTBITS	4 
#endif


#if WN == 210
#define RESTBITS	7
#endif

#define NDIGITS		(WK+1)
#define DIGITBITS	(WN/(NDIGITS))
#define BASE (1<<DIGITBITS)
#define NHASHES (2*BASE)
#define HASHESPERBLAKE (512/WN)
#define HASHLEN (WN+7)/8
#define HASHOUT (HASHESPERBLAKE*(HASHLEN))

// 2_log of number of buckets
#define BUCKBITS (DIGITBITS-RESTBITS)

#ifndef SAVEMEM
#if RESTBITS < 8
// can't save memory in such small buckets
#define SAVEMEM 1
#elif RESTBITS >= 8
// take advantage of law of large numbers (sum of 2^8 random numbers)
// this reduces (200,9) memory to under 144MB, with negligible discarding
#define SAVEMEM 9/14
#endif
#endif

// number of buckets
static const u32 NBUCKETS = 1 << BUCKBITS;
// bucket mask
static const u32 BUCKMASK = NBUCKETS - 1;
// 2_log of number of slots per bucket
static const u32 SLOTBITS = RESTBITS + 1 + 1;
static const u32 SLOTRANGE = 1 << SLOTBITS;
// number of slots per bucket
static const u32 NSLOTS = SLOTRANGE * SAVEMEM;
// SLOTBITS mask
static const u32 SLOTMASK = SLOTRANGE - 1;
// number of possible values of xhash (rest of n) bits
static const u32 NRESTS = 1 << RESTBITS;
// RESTBITS mask
static const u32 RESTMASK = NRESTS - 1;
// number of blocks of hashes extracted from single 512 bit blake2b output
static const u32 NBLOCKS = (NHASHES + HASHESPERBLAKE - 1) / HASHESPERBLAKE;
// nothing larger found in 100000 runs
static const u32 MAXSOLS = 10;











void setheader(blake2b_state *ctx, const unsigned char *header, const u32 headerLen, const unsigned char* nce, const u32 nonceLen) 
{
  uint32_t le_N = WN;
  uint32_t le_K = WK;
  uchar personal[] = "ZcashPoW01230123";
  memcpy(personal+8,  &le_N, 4);
  memcpy(personal+12, &le_K, 4);
  blake2b_param P[1];
  P->digest_length = HASHOUT;
  P->key_length    = 0;
  P->fanout        = 1;
  P->depth         = 1;
  P->leaf_length   = 0;
  P->node_offset   = 0;
  P->node_depth    = 0;
  P->inner_length  = 0;
  memset(P->reserved, 0, sizeof(P->reserved));
  memset(P->salt,     0, sizeof(P->salt));
  memcpy(P->personal, (const uint8_t *)personal, 16);
  eq_blake2b_init_param(ctx, P);
  eq_blake2b_update(ctx, (const uchar *)header, headerLen);
  eq_blake2b_update(ctx, (const uchar *)nce, nonceLen);
}




// tree node identifying its children as two different slots in
// a bucket on previous layer with the same rest bits (x-tra hash)
struct tree {
	u32 bid_s0_s1_x; // manual bitfields

	__device__ tree(const u32 idx, const u32 xh) {
		bid_s0_s1_x = idx << RESTBITS | xh;
	}
	__device__ tree(const u32 idx) {
		bid_s0_s1_x = idx;
	}

#ifdef XINTREE
  	__device__ tree(const u32 bid, const u32 s0, const u32 s1, const u32 xh) {
  	bid_s0_s1_x = ((((bid << SLOTBITS) | s0) << SLOTBITS) | s1) << RESTBITS | xh;
#else
  	__device__ tree(const u32 bid, const u32 s0, const u32 s1) {
  	bid_s0_s1_x = (((bid << SLOTBITS) | s0) << SLOTBITS) | s1;
#endif
	}
/*
	__device__ tree(const u32 bid, const u32 s0, const u32 s1, const u32 xh) {
#ifdef XINTREE
		bid_s0_s1_x = ((((bid << SLOTBITS) | s0) << SLOTBITS) | s1) << RESTBITS | xh;
#else
		bid_s0_s1_x = (((bid << SLOTBITS) | s0) << SLOTBITS) | s1;
#endif
	}
	*/

	__device__ u32 getindex() const {
#ifdef XINTREE
		return bid_s0_s1_x >> RESTBITS;
#else
		return bid_s0_s1_x;
#endif
	}
	__device__ u32 bucketid() const {
#ifdef XINTREE
		return bid_s0_s1_x >> (2 * SLOTBITS + RESTBITS);
#else
		return bid_s0_s1_x >> (2 * SLOTBITS);
#endif
	}
	__device__ u32 slotid0() const {
#ifdef XINTREE
		return (bid_s0_s1_x >> SLOTBITS + RESTBITS) & SLOTMASK;
#else
		return (bid_s0_s1_x >> SLOTBITS) & SLOTMASK;
#endif
	}
	__device__ u32 slotid1() const {
#ifdef XINTREE
		return (bid_s0_s1_x >> RESTBITS) & SLOTMASK;
#else
		return bid_s0_s1_x & SLOTMASK;
#endif
	}
	__device__ u32 xhash() const {
		return bid_s0_s1_x & RESTMASK;
	}
};

union hashunit {
	u32 word;
	uchar bytes[sizeof(u32)];
};

#define WORDS(bits)     ((bits + 31) / 32)
#define HASHWORDS0 WORDS(WN - DIGITBITS + RESTBITS)
#define HASHWORDS1 WORDS(WN - 2*DIGITBITS + RESTBITS)

struct slot0 {
	tree attr;
	hashunit hash[HASHWORDS0];
};

struct slot1 {
	tree attr;
	hashunit hash[HASHWORDS1]; 
};

// a bucket is NSLOTS treenodes
typedef slot0 bucket0[NSLOTS];
typedef slot1 bucket1[NSLOTS];
// the N-bit hash consists of K+1 n-bit "digits"
// each of which corresponds to a layer of NBUCKETS buckets
typedef bucket0 digit0[NBUCKETS];
typedef bucket1 digit1[NBUCKETS];

// size (in bytes) of hash in round 0 <= r < WK
u32 hhashsize(const u32 r) {
#ifdef XINTREE
	const u32 hashbits = WN - (r + 1) * DIGITBITS;
#else
	u32 hashbits; 
	if(WN == 210 && WK == 9){
		//Refer to the AION mining wiki for explanation of these hash size values
		switch(r){
			case 0:
				return 26;
			case 1:
				return 23;
			case 2:
				return 20;
			case 3:
				return 18;
			case 4:
				return 15;
			case 5:
				return 13;
			case 6:
				return 10;
			case 7:
				return 7;
			case 8:
				return 5;
			case 9:
				return 0;
			default:
				return 0;
		}
	}else{
		hashbits = 	WN - (r + 1) * DIGITBITS + RESTBITS;
	}
#endif
	return (hashbits + 7) / 8;
}
// size (in bytes) of hash in round 0 <= r < WK
__device__ u32 hashsize(const u32 r) {
#ifdef XINTREE
	const u32 hashbits = WN - (r + 1) * DIGITBITS;
#else
	u32 hashbits; 
	if(WN == 210 && WK == 9){
		//Refer to the AION mining wiki for explanation of these hash size values
		switch(r){
			case 0:
				return 26;
			case 1:
				return 23;
			case 2:
				return 20;
			case 3:
				return 18;
			case 4:
				return 15;
			case 5:
				return 13;
			case 6:
				return 10;
			case 7:
				return 7;
			case 8:
				return 5;
			case 9:
				return 0;
			default:
				return 0;
		}
	}else{
		hashbits = WN - (r + 1) * DIGITBITS + RESTBITS;
	}
#endif
	return (hashbits + 7) / 8;
}

u32 hhashwords(u32 bytes) {
	return (bytes + 3) / 4;
}

__device__ u32 hashwords(u32 bytes) {
	return (bytes + 3) / 4;
}

// manages hash and tree data
struct htalloc {
	bucket0 *trees0[(WK + 1) / 2];
	bucket1 *trees1[WK / 2];
};

typedef u32 bsizes[NBUCKETS];

struct equi210_9 {
	blake2b_state blake_ctx;
	htalloc hta;
	bsizes *nslots;
	proof *sols;
	u32 nsols;
	u32 nthreads;
	equi210_9(const u32 n_threads) {
		nthreads = n_threads;
	}
	void setheadernonce(unsigned char *header, const u32 len, unsigned char* nonce, const u32 nlen) {
		setheader(&blake_ctx, header, len, nonce, nlen);
		checkCudaErrors(hipMemset(nslots, 0, NBUCKETS * sizeof(u32)));
		nsols = 0;
	}
	__device__ u32 getnslots0(const u32 bid) {
		u32 &nslot = nslots[0][bid];
		const u32 n = min(nslot, NSLOTS);
		nslot = 0;
		return n;
	}
	__device__ u32 getnslots1(const u32 bid) {
		u32 &nslot = nslots[1][bid];
		const u32 n = min(nslot, NSLOTS);
		nslot = 0;
		return n;
	}
	__device__ void orderindices(u32 *indices, u32 size) {
		if (indices[0] > indices[size]) {
			for (u32 i = 0; i < size; i++) {
				const u32 tmp = indices[i];
				indices[i] = indices[size + i];
				indices[size + i] = tmp;
			}
		}
	}
	__device__ void listindices1(const tree t, u32 *indices) {
		const bucket0 &buck = hta.trees0[0][t.bucketid()];
		const u32 size = 1 << 0;
		indices[0] = buck[t.slotid0()].attr.getindex();
		indices[size] = buck[t.slotid1()].attr.getindex();
		orderindices(indices, size);
	}
	__device__ void listindices2(const tree t, u32 *indices) {
		const bucket1 &buck = hta.trees1[0][t.bucketid()];
		const u32 size = 1 << 1;
		listindices1(buck[t.slotid0()].attr, indices);
		listindices1(buck[t.slotid1()].attr, indices + size);
		orderindices(indices, size);
	}
	__device__ void listindices3(const tree t, u32 *indices) {
		const bucket0 &buck = hta.trees0[1][t.bucketid()];
		const u32 size = 1 << 2;
		listindices2(buck[t.slotid0()].attr, indices);
		listindices2(buck[t.slotid1()].attr, indices + size);
		orderindices(indices, size);
	}
	__device__ void listindices4(const tree t, u32 *indices) {
		const bucket1 &buck = hta.trees1[1][t.bucketid()];
		const u32 size = 1 << 3;
		listindices3(buck[t.slotid0()].attr, indices);
		listindices3(buck[t.slotid1()].attr, indices + size);
		orderindices(indices, size);
	}
	__device__ void listindices5(const tree t, u32 *indices) {
		const bucket0 &buck = hta.trees0[2][t.bucketid()];
		const u32 size = 1 << 4;
		listindices4(buck[t.slotid0()].attr, indices);
		listindices4(buck[t.slotid1()].attr, indices+size);
		orderindices(indices, size);
	}
	__device__ void listindices6(const tree t, u32 *indices) {
		const bucket1 &buck = hta.trees1[2][t.bucketid()];
		const u32 size = 1 << 5;
		listindices5(buck[t.slotid0()].attr, indices);
		listindices5(buck[t.slotid1()].attr, indices+size);
		orderindices(indices, size);
	}
	__device__ void listindices7(const tree t, u32 *indices) {
		const bucket0 &buck = hta.trees0[3][t.bucketid()];
		const u32 size = 1 << 6;
		listindices6(buck[t.slotid0()].attr, indices);
		listindices6(buck[t.slotid1()].attr, indices+size);
		orderindices(indices, size);
	}
	__device__ void listindices8(const tree t, u32 *indices) {
		const bucket1 &buck = hta.trees1[3][t.bucketid()];
		const u32 size = 1 << 7;
		listindices7(buck[t.slotid0()].attr, indices);
		listindices7(buck[t.slotid1()].attr, indices+size);
		orderindices(indices, size);
	}
	__device__ void listindices9(const tree t, u32 *indices) {
		const bucket0 &buck = hta.trees0[4][t.bucketid()];
		const u32 size = 1 << 8;
		listindices8(buck[t.slotid0()].attr, indices);
		listindices8(buck[t.slotid1()].attr, indices+size);
		orderindices(indices, size);
	}
	__device__ void candidate(const tree t) {
		proof prf;
#if WK==9
		listindices9(t, prf);
#elif WK==5
		listindices5(t, prf);
#else
#error not implemented
#endif
		if (probdupe(prf))
			return;
		u32 soli = atomicAdd(&nsols, 1);
		if (soli < MAXSOLS)
#if WK==9
			listindices9(t, sols[soli]);
#elif WK==5
			listindices5(t, sols[soli]);
#else
#error not implemented
#endif
	}
	void showbsizes(u32 r) {
#if defined(HIST) || defined(SPARK) || defined(LOGSPARK)
		u32 ns[NBUCKETS];
		checkCudaErrors(hipMemcpy(ns, nslots[r & 1], NBUCKETS * sizeof(u32), hipMemcpyDeviceToHost));
		u32 binsizes[65];
		memset(binsizes, 0, 65 * sizeof(u32));
		for (u32 bucketid = 0; bucketid < NBUCKETS; bucketid++) {
			u32 bsize = min(ns[bucketid], NSLOTS) >> (SLOTBITS - 6);
			binsizes[bsize]++;
		}
		for (u32 i = 0; i < 65; i++) {
#ifdef HIST
			printf(" %d:%d", i, binsizes[i]);
#else
#ifdef SPARK
			u32 sparks = binsizes[i] / SPARKSCALE;
#else
			u32 sparks = 0;
			for (u32 bs = binsizes[i]; bs; bs >>= 1) sparks++;
			sparks = sparks * 7 / SPARKSCALE;
#endif
			printf("\342\226%c", '\201' + sparks);
#endif
		}
		printf("\n");
#endif
		}
	// proper dupe test is a little costly on GPU, so allow false negatives
	__device__ bool probdupe(u32 *prf) {
		unsigned short susp[PROOFSIZE1];
		memset(susp, 0xffff, PROOFSIZE1 * sizeof(unsigned short));
		for (u32 i=0; i<PROOFSIZE1; i++) {
			u32 bin = prf[i] & (PROOFSIZE1-1);
			unsigned short msb = prf[i]>>WK;
			if (msb == susp[bin])
				return true;
			susp[bin] = msb;
		}
		return false;
	}
	struct htlayout {
		htalloc hta;
		u32 prevhashunits;
		u32 nexthashunits;
		u32 dunits;
		u32 prevbo;
		u32 nextbo;

		__device__ htlayout(equi210_9 *eq, u32 r) : hta(eq->hta), prevhashunits(0), dunits(0) {
			u32 nexthashbytes = hashsize(r);
			nexthashunits = hashwords(nexthashbytes);
			prevbo = 0;
			nextbo = nexthashunits * sizeof(hashunit) - nexthashbytes; // 0-3
			if (r) {
				u32 prevhashbytes = hashsize(r-1);
				prevhashunits = hashwords(prevhashbytes);
				prevbo = prevhashunits * sizeof(hashunit) - prevhashbytes; // 0-3
				dunits = prevhashunits - nexthashunits;
			}
		}
		__device__ u32 getxhash0(const slot0* pslot) const {
#ifdef XINTREE
			return pslot->attr.xhash();
#elif WN == 200 && RESTBITS == 4
			return pslot->hash->bytes[prevbo] >> 4;
#elif WN == 200 && RESTBITS == 8
			return (pslot->hash->bytes[prevbo] & 0xf) << 4 | pslot->hash->bytes[prevbo + 1] >> 4;
#elif WN == 144 && RESTBITS == 4
			return pslot->hash->bytes[prevbo] & 0xf;
#elif WN == 200 && RESTBITS == 6
			return (pslot->hash->bytes[prevbo] & 0x3) << 4 | pslot->hash->bytes[prevbo+1] >> 4;
#elif WN == 210 && RESTBITS == 7
			//Maintained for backwards compatibiilty
			return 0;
#else
#error non implemented
#endif
		}
		__device__ u32 getxhash1(const slot1* pslot) const {
#ifdef XINTREE
			return pslot->attr.xhash();
#elif WN == 200 && RESTBITS == 4
			return pslot->hash->bytes[prevbo] & 0xf;
#elif WN == 200 && RESTBITS == 8
			return pslot->hash->bytes[prevbo];
#elif WN == 144 && RESTBITS == 4
			return pslot->hash->bytes[prevbo] & 0xf;
#elif WN == 200 && RESTBITS == 6
			return pslot->hash->bytes[prevbo] & 0x3f;
#elif WN == 210 && RESTBITS == 7
			//Maintained for backwards compatibiilty
			return 0;
#else
#error non implemented
#endif
		}
		__device__ bool equal(const hashunit *hash0, const hashunit *hash1) const {
			return hash0[prevhashunits - 1].word == hash1[prevhashunits - 1].word;
		}

		__device__ bool equal_210(const slot0 *pslot0, const slot0 *pslot1) const {
			//Check last 21 bits for collision
			return ((pslot0->hash->bytes[prevbo + 1] & 0x7) == (pslot1->hash->bytes[prevbo + 1] & 0x7) &&
				    (pslot0->hash->bytes[prevbo + 2]) == (pslot1->hash->bytes[prevbo + 2]) &&
				    (pslot0->hash->bytes[prevbo + 3]) == (pslot1->hash->bytes[prevbo + 3]) &&
				    (pslot0->hash->bytes[prevbo + 4] >> 6) == (pslot1->hash->bytes[prevbo + 4] >> 6));
		}
	};

	struct collisiondata {
#ifdef XBITMAP
#if NSLOTS > 64
#error cant use XBITMAP with more than 64 slots
#endif
		u64 xhashmap[NRESTS];
		u64 xmap;
#else
#if RESTBITS <= 6
		typedef uchar xslot;
#else
		typedef u16 xslot;
#endif
		static const xslot xnil = ~0;
		xslot xhashslots[NRESTS];
		xslot nextxhashslot[NSLOTS];
		xslot nextslot;
#endif
		u32 s0;

		__device__ void clear() {
#ifdef XBITMAP
			memset(xhashmap, 0, NRESTS * sizeof(u64));
#else
			memset(xhashslots, xnil, NRESTS * sizeof(xslot));
			memset(nextxhashslot, xnil, NSLOTS * sizeof(xslot));
#endif
		}
		__device__ bool addslot(u32 s1, u32 xh) {
#ifdef XBITMAP
			xmap = xhashmap[xh];
			xhashmap[xh] |= (u64)1 << s1;
			s0 = ~0;
			return true;
#else
			nextslot = xhashslots[xh];
			nextxhashslot[s1] = nextslot;
			xhashslots[xh] = s1;
			return true;
#endif
		}
		__device__ bool nextcollision() const {
#ifdef XBITMAP
			return xmap != 0;
#else
			return nextslot != xnil;
#endif
		}
		__device__ u32 slot() {
#ifdef XBITMAP
			const u32 ffs = __ffsll(xmap);
			s0 += ffs; xmap >>= ffs;
#else
			nextslot = nextxhashslot[s0 = nextslot];
#endif
			return s0;
		}
	};
		};

__global__ void digitH(equi210_9 *eq) {
	uchar hash[HASHOUT];
	blake2b_state state;
	equi210_9::htlayout htl(eq, 0);
	const u32 hashbytes = hashsize(0); // always 23 ?
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 block = id; block < NBLOCKS; block += eq->nthreads) {
		state = eq->blake_ctx;
		blake2b_gpu_hash(&state, block, hash, HASHOUT);
		for (u32 i = 0; i<HASHESPERBLAKE; i++) {
			const uchar *ph = hash + i * HASHLEN;
#if BUCKBITS == 16 && RESTBITS == 4
			const u32 bucketid = ((u32)ph[0] << 8) | ph[1];
#ifdef XINTREE
			const u32 xhash = ph[2] >> 4;
#endif
#elif BUCKBITS == 14 && RESTBITS == 6
			const u32 bucketid = ((u32)ph[0] << 6) | ph[1] >> 2;
#elif BUCKBITS == 12 && RESTBITS == 8
			const u32 bucketid = ((u32)ph[0] << 4) | ph[1] >> 4;
#elif BUCKBITS == 20 && RESTBITS == 4
			const u32 bucketid = ((((u32)ph[0] << 8) | ph[1]) << 4) | ph[2] >> 4;
#ifdef XINTREE
			const u32 xhash = ph[2] & 0xf;
#endif
#elif BUCKBITS == 12 && RESTBITS == 4
			const u32 bucketid = ((u32)ph[0] << 4) | ph[1] >> 4;
			const u32 xhash = ph[1] & 0xf;
#elif BUCKBITS == 14 && RESTBITS == 7
      		const u32 bucketid = ((u32)ph[0] << (BUCKBITS - 8)) | ph[1] >> (16 - BUCKBITS);
#else
#error not implemented
#endif
			const u32 slot = atomicAdd(&eq->nslots[0][bucketid], 1);
			if (slot >= NSLOTS)
				continue;
			slot0 &s = eq->hta.trees0[0][bucketid][slot];
#ifdef XINTREE
			s.attr = tree(block*HASHESPERBLAKE+i, xhash);
#else
			s.attr = tree(block*HASHESPERBLAKE+i);
#endif
			memcpy(s.hash->bytes+htl.nextbo, ph+HASHLEN-hashbytes, hashbytes);
		}
	}
}

__global__ void digitO(equi210_9 *eq, const u32 r) {
	equi210_9::htlayout htl(eq, r);
	equi210_9::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot0 *buck = htl.hta.trees0[(r - 1) / 2][bucketid];
		u32 bsize = eq->getnslots0(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot0 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash0(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot0 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				u32 xorbucketid;
				u32 xhash;
				const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;
#if WN == 200 && BUCKBITS == 16 && RESTBITS == 4 && defined(XINTREE)
				xorbucketid = ((((u32)(bytes0[htl.prevbo] ^ bytes1[htl.prevbo]) & 0xf) << 8)
					| (bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1])) << 4
					| (xhash = bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 4;
				xhash &= 0xf;
#elif WN == 144 && BUCKBITS == 20 && RESTBITS == 4
				xorbucketid = ((((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) << 8)
					| (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2])) << 4)
					| (xhash = bytes0[htl.prevbo + 3] ^ bytes1[htl.prevbo + 3]) >> 4;
				xhash &= 0xf;
#elif WN == 96 && BUCKBITS == 12 && RESTBITS == 4
				xorbucketid = ((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) << 4)
					| (xhash = bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 4;
				xhash &= 0xf;
#elif WN == 200 && BUCKBITS == 14 && RESTBITS == 6
				xorbucketid = ((((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) & 0xf) << 8)
					| (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2])) << 2
					| (bytes0[htl.prevbo + 3] ^ bytes1[htl.prevbo + 3]) >> 6;
#elif WN == 210 && BUCKBITS == 14 && RESTBITS == 7
				//Included to maintain backwards compatibility
				xorbucketid = 0;
				xhash = 0;
#else
#error not implemented
#endif
				const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot1 &xs = htl.hta.trees1[r/2][xorbucketid][xorslot];
#ifdef XINTREE
				xs.attr = tree(bucketid, s0, s1, xhash);
#else
				xs.attr = tree(bucketid, s0, s1);
#endif
				for (u32 i=htl.dunits; i < htl.prevhashunits; i++)
					xs.hash[i - htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
			}
		}
	}
}

__global__ void digitE(equi210_9 *eq, const u32 r) {
	equi210_9::htlayout htl(eq, r);
	equi210_9::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot1 *buck = htl.hta.trees1[(r - 1) / 2][bucketid];
		u32 bsize = eq->getnslots1(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot1 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash1(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot1 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				u32 xorbucketid;
				u32 xhash;
				const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;
#if WN == 200 && BUCKBITS == 16 && RESTBITS == 4 && defined(XINTREE)
				xorbucketid = ((u32)(bytes0[htl.prevbo] ^ bytes1[htl.prevbo]) << 8)
					| (bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]);
				xhash = (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 4;
#elif WN == 144 && BUCKBITS == 20 && RESTBITS == 4
				xorbucketid = ((((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) << 8)
					| (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2])) << 4)
					| (bytes0[htl.prevbo + 3] ^ bytes1[htl.prevbo + 3]) >> 4;
#elif WN == 96 && BUCKBITS == 12 && RESTBITS == 4
				xorbucketid = ((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) << 4)
					| (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 4;
#elif WN == 200 && BUCKBITS == 14 && RESTBITS == 6
				xorbucketid = ((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) << 6)
					| (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 2;
#elif WN == 210 && BUCKBITS == 14 && RESTBITS == 7
				//Included to maintain backwards compatibility
				xorbucketid = 0;
				xhash = 0;
#else
#error not implemented
#endif
				const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot0 &xs = htl.hta.trees0[r / 2][xorbucketid][xorslot];
#ifdef XINTREE
				xs.attr = tree(bucketid, s0, s1, xhash);
#else
				xs.attr = tree(bucketid, s0, s1);
#endif
				for (u32 i = htl.dunits; i < htl.prevhashunits; i++)
					xs.hash[i - htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
			}
		}
	}
}

#ifdef UNROLL
__global__ void digit_1(equi210_9 *eq) {
	equi210_9::htlayout htl(eq, 1);
	equi210_9::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot0 *buck = htl.hta.trees0[0][bucketid];
		u32 bsize = eq->getnslots0(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot0 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash0(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot0 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 bexor = __byte_perm(xor0, 0, 0x0123);
				const u32 xorbucketid = bexor >> 4 & BUCKMASK;
				const u32 xhash = bexor & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot1 &xs = htl.hta.trees1[0][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
				xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
				xs.hash[2].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
				xs.hash[3].word = pslot0->hash[4].word ^ pslot1->hash[4].word;
				xs.hash[4].word = pslot0->hash[5].word ^ pslot1->hash[5].word;
			}
		}
	}
}
__global__ void digit2(equi210_9 *eq) {
	equi210_9::htlayout htl(eq, 2);
	equi210_9::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot1 *buck = htl.hta.trees1[0][bucketid];
		u32 bsize = eq->getnslots1(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot1 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash1(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot1 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 bexor = __byte_perm(xor0, 0, 0x0123);
				const u32 xorbucketid = bexor >> 16;
				const u32 xhash = bexor >> 12 & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot0 &xs = htl.hta.trees0[1][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = xor0;
				xs.hash[1].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
				xs.hash[2].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
				xs.hash[3].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
				xs.hash[4].word = pslot0->hash[4].word ^ pslot1->hash[4].word;
			}
		}
	}
}
__global__ void digit3(equi210_9 *eq) {
	equi210_9::htlayout htl(eq, 3);
	equi210_9::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot0 *buck = htl.hta.trees0[1][bucketid];
		u32 bsize = eq->getnslots0(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot0 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash0(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot0 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
				const u32 bexor = __byte_perm(xor0, xor1, 0x1234);
				const u32 xorbucketid = bexor >> 4 & BUCKMASK;
				const u32 xhash = bexor & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot1 &xs = htl.hta.trees1[1][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = xor1;
				xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
				xs.hash[2].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
				xs.hash[3].word = pslot0->hash[4].word ^ pslot1->hash[4].word;
			}
		}
	}
}
__global__ void digit4(equi210_9 *eq) {
	equi210_9::htlayout htl(eq, 4);
	equi210_9::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot1 *buck = htl.hta.trees1[1][bucketid];
		u32 bsize = eq->getnslots1(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot1 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash1(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot1 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 bexor = __byte_perm(xor0, 0, 0x4123);
				const u32 xorbucketid = bexor >> 8;
				const u32 xhash = bexor >> 4 & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot0 &xs = htl.hta.trees0[2][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = xor0;
				xs.hash[1].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
				xs.hash[2].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
				xs.hash[3].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
			}
		}
	}
}
__global__ void digit5(equi210_9 *eq) {
	equi210_9::htlayout htl(eq, 5);
	equi210_9::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot0 *buck = htl.hta.trees0[2][bucketid];
		u32 bsize = eq->getnslots0(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot0 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash0(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot0 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
				const u32 bexor = __byte_perm(xor0, xor1, 0x2345);
				const u32 xorbucketid = bexor >> 4 & BUCKMASK;
				const u32 xhash = bexor & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot1 &xs = htl.hta.trees1[2][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = xor1;
				xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
				xs.hash[2].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
			}
		}
	}
}
__global__ void digit6(equi210_9 *eq) {
	equi210_9::htlayout htl(eq, 6);
	equi210_9::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot1 *buck = htl.hta.trees1[2][bucketid];
		u32 bsize = eq->getnslots1(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot1 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash1(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot1 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
				const u32 bexor = __byte_perm(xor0, xor1, 0x2345);
				const u32 xorbucketid = bexor >> 16;
				const u32 xhash = bexor >> 12 & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot0 &xs = htl.hta.trees0[3][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = xor1;
				xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
			}
		}
	}
}
__global__ void digit7(equi210_9 *eq) { 
	equi210_9::htlayout htl(eq, 7);
	equi210_9::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot0 *buck = htl.hta.trees0[3][bucketid];
		u32 bsize = eq->getnslots0(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot0 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash0(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot0 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 bexor = __byte_perm(xor0, 0, 0x4012);
				const u32 xorbucketid = bexor >> 4 & BUCKMASK;
				const u32 xhash = bexor & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot1 &xs = htl.hta.trees1[3][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = xor0;
				xs.hash[1].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
			}
		}
	}
}
__global__ void digit8(equi210_9 *eq) {
	equi210_9::htlayout htl(eq, 8);
	equi210_9::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot1 *buck = htl.hta.trees1[3][bucketid];
		u32 bsize = eq->getnslots1(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot1 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash1(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot1 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
				const u32 bexor = __byte_perm(xor0, xor1, 0x3456);
				const u32 xorbucketid = bexor >> 16;
				const u32 xhash = bexor >> 12 & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot0 &xs = htl.hta.trees0[4][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = xor1;
			}
		}
	}
}
#endif


#if WN == 210 && WK == 9
__global__ void digit_1(equi210_9 *eq, const u32 r) {
  equi210_9::htlayout htl(eq, r);
  equi210_9::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[(r-1)/2][bucketid];
    u32 bsize = eq->getnslots0(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      u32 xhash0 = (pslot1->hash->bytes[htl.prevbo] & 0x3) << 5 | (pslot1->hash->bytes[htl.prevbo+1] >> 3);
      if(!cd.addslot(s1, xhash0))
	  	continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
        	continue;
        u32 xorbucketid;
        const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;

        xorbucketid = ((((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) & 0x7) << 8) | (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2])) << 3 | (bytes0[htl.prevbo + 3] ^ bytes1[htl.prevbo + 3]) >> 5;

        const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot1 &xs = htl.hta.trees1[r/2][xorbucketid][xorslot];

        xs.attr = tree(bucketid, s0, s1);

        for (u32 i=htl.dunits; i < htl.prevhashunits; i++)
          xs.hash[i-htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
      }
    }
  }
}

__global__ void digit_2(equi210_9 *eq, const u32 r) {
  equi210_9::htlayout htl(eq, r);
  equi210_9::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot1 *buck = htl.hta.trees1[(r-1)/2][bucketid];
    u32 bsize = eq->getnslots1(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot1 *pslot1 = buck + s1;
      u32 xhash1 = (pslot1->hash->bytes[htl.prevbo] & 0x1f) << 2 | (pslot1->hash->bytes[htl.prevbo+1] >> 6);
		if(!cd.addslot(s1, xhash1))
	  		continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot1 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
        	continue;
        u32 xorbucketid;
        const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;

        xorbucketid = ((((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) & 0x3f) << 8) | ((bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2])));

        const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot0 &xs = htl.hta.trees0[r/2][xorbucketid][xorslot];

        xs.attr = tree(bucketid, s0, s1);

        for (u32 i=htl.dunits; i < htl.prevhashunits; i++){
          xs.hash[i-htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
        }
      }
    }
  }
}

__global__ void digit_3(equi210_9 *eq, const u32 r) {
  equi210_9::htlayout htl(eq, r);
  equi210_9::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[(r-1)/2][bucketid];
    u32 bsize = eq->getnslots0(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      u32 xhash0 = pslot1->hash->bytes[htl.prevbo] >>1;
      if(!cd.addslot(s1, xhash0))
	  	continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
        	continue;
        u32 xorbucketid;
        const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;

        xorbucketid = ((((u32)(bytes0[htl.prevbo] ^ bytes1[htl.prevbo]) & 0x1) << 8) | (bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1])) << 5 | ((bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 3);

        const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot1 &xs = htl.hta.trees1[r/2][xorbucketid][xorslot];

        xs.attr = tree(bucketid, s0, s1);

        for (u32 i=htl.dunits; i < htl.prevhashunits; i++)
          xs.hash[i-htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
      }
    }
  }
}

__global__ void digit_4(equi210_9 *eq, const u32 r) {
  equi210_9::htlayout htl(eq, r);
  equi210_9::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot1 *buck = htl.hta.trees1[(r-1)/2][bucketid];
    u32 bsize = eq->getnslots1(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot1 *pslot1 = buck + s1;
      u32 xhash1 = (pslot1->hash->bytes[htl.prevbo] & 0x7) << 4 | (pslot1->hash->bytes[htl.prevbo+1] >> 4);
		if(!cd.addslot(s1, xhash1))
	  		continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot1 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
        	continue;
        
        u32 xorbucketid;
        const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;
        xorbucketid = (((((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) & 0xf)) << 8) | (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2])) << 2 | ((bytes0[htl.prevbo + 3] ^ bytes1[htl.prevbo + 3]) >> 6);

        const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot0 &xs = htl.hta.trees0[r/2][xorbucketid][xorslot];

        xs.attr = tree(bucketid, s0, s1);

        for (u32 i=htl.dunits; i < htl.prevhashunits; i++){
          xs.hash[i-htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
        }
      }
    }
  }
}

__global__ void digit_5(equi210_9 *eq, const u32 r) {
  equi210_9::htlayout htl(eq, r);
  equi210_9::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[(r-1)/2][bucketid];
    u32 bsize = eq->getnslots0(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      u32 xhash0 = (pslot1->hash->bytes[htl.prevbo] & 0x3f) << 1 | (pslot1->hash->bytes[htl.prevbo+1] >> 7);
	  	if(!cd.addslot(s1, xhash0))
	  		continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
        	continue;
        
        u32 xorbucketid;
        const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;
       
	    xorbucketid = (((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) & 0x7f) << 7) | (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 1;

        const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot1 &xs = htl.hta.trees1[r/2][xorbucketid][xorslot];

        xs.attr = tree(bucketid, s0, s1);

        for (u32 i=htl.dunits; i < htl.prevhashunits; i++){
          xs.hash[i-htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
        }
      }
    }
  }
}

__global__ void digit_6(equi210_9 *eq, const u32 r) {
  equi210_9::htlayout htl(eq, r);
  equi210_9::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot1 *buck = htl.hta.trees1[(r-1)/2][bucketid];
    u32 bsize = eq->getnslots1(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot1 *pslot1 = buck + s1;
      u32 xhash1 = (pslot1->hash->bytes[htl.prevbo] & 0x1) << 6 | (pslot1->hash->bytes[htl.prevbo+1] >> 2);
		if(!cd.addslot(s1, xhash1))
	  		continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot1 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
        	continue;
        
        u32 xorbucketid;
        const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;

        xorbucketid = ((((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) & 0x3) << 8) | (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2])) << 4 | (bytes0[htl.prevbo + 3] ^ bytes1[htl.prevbo + 3]) >> 4;

        const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot0 &xs = htl.hta.trees0[r/2][xorbucketid][xorslot];

        xs.attr = tree(bucketid, s0, s1);

        for (u32 i=htl.dunits; i < htl.prevhashunits; i++){
          xs.hash[i-htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
        }
      }
    }
  }
}

__global__ void digit_7(equi210_9 *eq, const u32 r) {
  equi210_9::htlayout htl(eq, r);
  equi210_9::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[(r-1)/2][bucketid];
    u32 bsize = eq->getnslots0(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      u32 xhash0 = (pslot1->hash->bytes[htl.prevbo] & 0xf) << 3 | (pslot1->hash->bytes[htl.prevbo+1] >> 5);
	  	if(!cd.addslot(s1, xhash0))
	  		continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
        	continue;
        
        u32 xorbucketid;
        const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;

        xorbucketid = ((((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) & 0x1f) << 8) | (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2])) << 1 | (bytes0[htl.prevbo + 3] ^ bytes1[htl.prevbo + 3]) >> 7;

        const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot1 &xs = htl.hta.trees1[r/2][xorbucketid][xorslot];

        xs.attr = tree(bucketid, s0, s1);

        for (u32 i=htl.dunits; i < htl.prevhashunits; i++){
          xs.hash[i-htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
        }
      }
    }
  }
}

__global__ void digit_8(equi210_9 *eq, const u32 r) {
  equi210_9::htlayout htl(eq, r);
  equi210_9::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot1 *buck = htl.hta.trees1[(r-1)/2][bucketid];
    u32 bsize = eq->getnslots1(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot1 *pslot1 = buck + s1;
      u32 xhash1 = (pslot1->hash->bytes[htl.prevbo] & 0x7f);
		if(!cd.addslot(s1, xhash1))
	  		continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot1 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
        	continue;
        
        u32 xorbucketid;
        const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;

        xorbucketid = ((u32)(bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]) << 6) | (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 2;

        const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot0 &xs = htl.hta.trees0[r/2][xorbucketid][xorslot];

        xs.attr = tree(bucketid, s0, s1);

        for (u32 i=htl.dunits; i < htl.prevhashunits; i++){
          xs.hash[i-htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
        }
      }
    }
  }
}

__global__ void digit_9(equi210_9 *eq) {
  equi210_9::collisiondata cd;
  equi210_9::htlayout htl(eq, WK);
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[(WK-1)/2][bucketid];
    u32 bsize = eq->getnslots0(bucketid); // assume WK odd
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      u32 xhash0 = ((pslot1->hash->bytes[htl.prevbo] & 0x3) << 5) | ((pslot1->hash->bytes[htl.prevbo+1]) >> 3);
      cd.addslot(s1, xhash0);
      for (; cd.nextcollision(); ) { // assume WK odd
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;

        //Check last 21 bits for collision, disjoint and process candidate.

        if (htl.equal_210(pslot0,pslot1)) {
          eq->candidate(tree(bucketid, s0, s1));
        }
      }
    }
  }
}
#endif


__global__ void digitK(equi210_9 *eq) {
	equi210_9::collisiondata cd;
	equi210_9::htlayout htl(eq, WK);
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot0 *buck = htl.hta.trees0[(WK - 1) / 2][bucketid];
		u32 bsize = eq->getnslots0(bucketid); // assume WK odd
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot0 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash0(pslot1))) // assume WK odd
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot0 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash)) {
#ifdef XINTREE
					eq->candidate(tree(bucketid, s0, s1, 0));
#else
					eq->candidate(tree(bucketid, s0, s1));
#endif
				}
			}
		}
	}
}


eq_cuda_context210_9::eq_cuda_context210_9(int thrid, int devid, fn_validate validate, fn_cancel cancel)	
{
    threadsperblock = 64;
    device_id = devid;
    thread_id = thrid;

    m_fnValidate = validate;
    m_fnCancel = cancel;

    hipDeviceProp_t device_props;
    checkCudaErrors(hipGetDeviceProperties(&device_props, device_id));
    totalblocks = device_props.multiProcessorCount * 7;
    
	eq = new equi210_9(threadsperblock * totalblocks);
	sol_memory = malloc(sizeof(proof) * MAXSOLS + 4096);
	solutions = (proof*)(((long long)sol_memory + 4095) & -4096);

	checkCudaErrors(hipSetDevice(device_id));
	checkCudaErrors(hipDeviceReset());
	checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
	checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

	checkCudaErrors(hipMalloc((void**)&heap0, sizeof(digit0)));
	checkCudaErrors(hipMalloc((void**)&heap1, sizeof(digit1)));
	for (u32 r = 0; r < WK; r++)
		if ((r & 1) == 0)
			eq->hta.trees0[r / 2] = (bucket0 *)(heap0 + r / 2);
		else
			eq->hta.trees1[r / 2] = (bucket1 *)(heap1 + r / 2);

	checkCudaErrors(hipMalloc((void**)&eq->nslots, 2 * NBUCKETS * sizeof(u32)));
	checkCudaErrors(hipMalloc((void**)&eq->sols, MAXSOLS * sizeof(proof)));

	checkCudaErrors(hipMalloc((void**)&device_eq, sizeof(equi210_9)));
}


eq_cuda_context210_9::~eq_cuda_context210_9()
{
	/*checkCudaErrors(hipFree(eq->nslots));
	checkCudaErrors(hipFree(eq->sols));
	checkCudaErrors(hipFree(eq->hta.trees0[0]));
	checkCudaErrors(hipFree(eq->hta.trees1[0]));*/
	checkCudaErrors(hipSetDevice(device_id));
	checkCudaErrors(hipDeviceReset());
	free(sol_memory);
	delete eq;
}

std::vector<unsigned char> GetMinimalFromIndices(std::vector<uint32_t> indices,
                                                 size_t cBitLen);

bool eq_cuda_context210_9::solve(unsigned char *pblock, unsigned char *header, unsigned int headerlen)
{
	checkCudaErrors(hipSetDevice(device_id)); 

	eq->setheadernonce(header, headerlen-32, header+headerlen-32, 32);
	checkCudaErrors(hipMemcpy(device_eq, eq, sizeof(equi210_9), hipMemcpyHostToDevice));

	digitH << <totalblocks, threadsperblock >> >(device_eq);
	if (m_fnCancel()) return false;
#if BUCKBITS == 16 && RESTBITS == 4 && defined XINTREE && defined(UNROLL)
	digit_1 << <totalblocks, threadsperblock >> >(device_eq);
	if (m_fnCancel()) return false;
	digit2 << <totalblocks, threadsperblock >> >(device_eq);
	if (m_fnCancel()) return false;
	digit3 << <totalblocks, threadsperblock >> >(device_eq);
	if (m_fnCancel()) return false;
	digit4 << <totalblocks, threadsperblock >> >(device_eq);
	if (m_fnCancel()) return false;
	digit5 << <totalblocks, threadsperblock >> >(device_eq);
	if (m_fnCancel()) return false;
	digit6 << <totalblocks, threadsperblock >> >(device_eq);
	if (m_fnCancel()) return false;
	digit7 << <totalblocks, threadsperblock >> >(device_eq);
	if (m_fnCancel()) return false;
	digit8 << <totalblocks, threadsperblock >> >(device_eq);
	if (m_fnCancel()) return false;
	digitK << <totalblocks, threadsperblock >> >(device_eq);

#elif BUCKBITS == 14 && RESTBITS == 7
	digit_1 << <totalblocks, threadsperblock >> >(device_eq, 1);
	if (m_fnCancel()) return false;
	digit_2 << <totalblocks, threadsperblock >> >(device_eq, 2);
	if (m_fnCancel()) return false;
	digit_3 << <totalblocks, threadsperblock >> >(device_eq, 3);
	if (m_fnCancel()) return false;
	digit_4 << <totalblocks, threadsperblock >> >(device_eq, 4);
	if (m_fnCancel()) return false;
	digit_5 << <totalblocks, threadsperblock >> >(device_eq, 5);
	if (m_fnCancel()) return false;
	digit_6 << <totalblocks, threadsperblock >> >(device_eq, 6);
	if (m_fnCancel()) return false;
	digit_7 << <totalblocks, threadsperblock >> >(device_eq, 7);
	if (m_fnCancel()) return false;
	digit_8 << <totalblocks, threadsperblock >> >(device_eq, 8);
	if (m_fnCancel()) return false;
	digit_9 << <totalblocks, threadsperblock >> >(device_eq);
#else
	for (u32 r = 1; r < WK; r++) {
		r & 1 ? digitO << <totalblocks, threadsperblock >> >(device_eq, r)
			: digitE << <totalblocks, threadsperblock >> >(device_eq, r);
	}

	if (m_fnCancel()) return false;
	digitK << <totalblocks, threadsperblock >> >(device_eq);
#endif

	checkCudaErrors(hipMemcpy(eq, device_eq, sizeof(equi210_9), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(solutions, eq->sols, MAXSOLS * sizeof(proof), hipMemcpyDeviceToHost));

	for (unsigned s = 0; (s < eq->nsols) && (s < MAXSOLS); s++)
	{
		std::vector<uint32_t> index_vector(PROOFSIZE1);
		for (u32 i = 0; i < PROOFSIZE1; i++) {
			index_vector[i] = solutions[s][i];
		}

		std::vector<unsigned char> sol_char = GetMinimalFromIndices(index_vector, DIGITBITS);
		if (m_fnValidate(sol_char, pblock, thread_id)) 
        {
             // If we find a POW solution, do not try other solutions
             // because they become invalid as we created a new block in blockchain.
             return true;
        }        
	}
	return false;
}