#include "hip/hip_runtime.h"
// Blake2-B CUDA Implementation
// tpruvot@github July 2016
// permission granted to use under MIT license
// modified for use in Zcash by John Tromp September 2016

#include "blake2/blake2.h"
typedef uint32_t u32;
typedef uint16_t u16;
typedef uint8_t u8;
typedef uint8_t uchar;

/**
 * uint2 direct ops by c++ operator definitions
 */
static __device__ __forceinline__ uint2 operator^ (uint2 a, uint2 b) {
  return make_uint2(a.x ^ b.x, a.y ^ b.y);
}
// uint2 ROR/ROL methods
__device__ __forceinline__ uint2 ROR2(const uint2 a, const int offset) {
  uint2 result;
#if __CUDA_ARCH__ > 300
  if (offset < 32) {
          asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.x), "r"(a.y), "r"(offset));
          asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.y), "r"(a.x), "r"(offset));
  } else /* if (offset < 64) */ {
          /* offset SHOULD BE < 64 ! */
          asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
          asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
  }
#else
  if (!offset)
          result = a;
  else if (offset < 32) {
          result.y = ((a.y >> offset) | (a.x << (32 - offset)));
          result.x = ((a.x >> offset) | (a.y << (32 - offset)));
  } else if (offset == 32) {
          result.y = a.x;
          result.x = a.y;
  } else {
          result.y = ((a.x >> (offset - 32)) | (a.y << (64 - offset)));
          result.x = ((a.y >> (offset - 32)) | (a.x << (64 - offset)));
  }
#endif
  return result;
}
__device__ __forceinline__ uint2 SWAPUINT2(uint2 value) {
  return make_uint2(value.y, value.x);
}
#ifdef __CUDA_ARCH__
__device__ __inline__ uint2 ROR24(const uint2 a) {
  uint2 result;
  result.x = __byte_perm(a.y, a.x, 0x2107);
  result.y = __byte_perm(a.y, a.x, 0x6543);
  return result;
}
__device__ __inline__ uint2 ROR16(const uint2 a) {
  uint2 result;
  result.x = __byte_perm(a.y, a.x, 0x1076);
  result.y = __byte_perm(a.y, a.x, 0x5432);
  return result;
}
#else
#define ROR24(u) ROR2(u,24)
#define ROR16(u) ROR2(u,16)
#endif

typedef uint64_t u64;

static __constant__ const int8_t blake2b_sigma[12][16] = {
  { 0,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15 } ,
  { 14, 10, 4,  8,  9,  15, 13, 6,  1,  12, 0,  2,  11, 7,  5,  3  } ,
  { 11, 8,  12, 0,  5,  2,  15, 13, 10, 14, 3,  6,  7,  1,  9,  4  } ,
  { 7,  9,  3,  1,  13, 12, 11, 14, 2,  6,  5,  10, 4,  0,  15, 8  } ,
  { 9,  0,  5,  7,  2,  4,  10, 15, 14, 1,  11, 12, 6,  8,  3,  13 } ,
  { 2,  12, 6,  10, 0,  11, 8,  3,  4,  13, 7,  5,  15, 14, 1,  9  } ,
  { 12, 5,  1,  15, 14, 13, 4,  10, 0,  7,  6,  3,  9,  2,  8,  11 } ,
  { 13, 11, 7,  14, 12, 1,  3,  9,  5,  0,  15, 4,  8,  6,  2,  10 } ,
  { 6,  15, 14, 9,  11, 3,  0,  8,  12, 2,  13, 7,  1,  4,  10, 5  } ,
  { 10, 2,  8,  4,  7,  6,  1,  5,  15, 11, 9,  14, 3,  12, 13, 0  } ,
  { 0,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15 } ,
  { 14, 10, 4,  8,  9,  15, 13, 6,  1,  12, 0,  2,  11, 7,  5,  3  }
};

__device__ __forceinline__
static void G(const int r, const int i, u64 &a, u64 &b, u64 &c, u64 &d, u64 const m[16]) {
  a = a + b + m[ blake2b_sigma[r][2*i] ];
  ((uint2*)&d)[0] = SWAPUINT2( ((uint2*)&d)[0] ^ ((uint2*)&a)[0] );
  c = c + d;
  ((uint2*)&b)[0] = ROR24( ((uint2*)&b)[0] ^ ((uint2*)&c)[0] );
  a = a + b + m[ blake2b_sigma[r][2*i+1] ];
  ((uint2*)&d)[0] = ROR16( ((uint2*)&d)[0] ^ ((uint2*)&a)[0] );
  c = c + d;
  ((uint2*)&b)[0] = ROR2( ((uint2*)&b)[0] ^ ((uint2*)&c)[0], 63U);
}

#define ROUND(r) \
  G(r, 0, v[0], v[4], v[ 8], v[12], m); \
  G(r, 1, v[1], v[5], v[ 9], v[13], m); \
  G(r, 2, v[2], v[6], v[10], v[14], m); \
  G(r, 3, v[3], v[7], v[11], v[15], m); \
  G(r, 4, v[0], v[5], v[10], v[15], m); \
  G(r, 5, v[1], v[6], v[11], v[12], m); \
  G(r, 6, v[2], v[7], v[ 8], v[13], m); \
  G(r, 7, v[3], v[4], v[ 9], v[14], m);

__device__ void blake2b_gpu_hash(blake2b_state *state, u32 idx, uchar *hash, const u32 outlen) {
  const u32 leb = htole32(idx);
  memcpy(state->buf + state->buflen, &leb, sizeof(u32));
  state->buflen += sizeof(u32);
  state->counter += state->buflen;
  memset(state->buf + state->buflen, 0, BLAKE2B_BLOCKBYTES - state->buflen);

  u64 *d_data = (u64 *)state->buf;
  u64 m[16];

  m[0] = d_data[0];
  m[1] = d_data[1];
  m[2] = d_data[2];
  m[3] = d_data[3];
  m[4] = d_data[4];
  m[5] = d_data[5];
  m[6] = d_data[6];
  m[7] = d_data[7];
  m[8] = d_data[8];
  m[9] = d_data[9];
  m[10] = d_data[10];
  m[11] = d_data[11];
  m[12] = d_data[12];
  m[13] = d_data[13];
  m[14] = d_data[14];
  m[15] = d_data[15];

  u64 v[16];

  v[0] = state->h[0];
  v[1] = state->h[1];
  v[2] = state->h[2];
  v[3] = state->h[3];
  v[4] = state->h[4];
  v[5] = state->h[5];
  v[6] = state->h[6];
  v[7] = state->h[7];
  v[8] = 0x6a09e667f3bcc908;
  v[9] = 0xbb67ae8584caa73b;
  v[10] = 0x3c6ef372fe94f82b;
  v[11] = 0xa54ff53a5f1d36f1;
  v[12] = 0x510e527fade682d1 ^ state->counter;
  v[13] = 0x9b05688c2b3e6c1f;
  v[14] = 0x1f83d9abfb41bd6b ^ 0xffffffffffffffff;
  v[15] = 0x5be0cd19137e2179;

  ROUND( 0 );
  ROUND( 1 );
  ROUND( 2 );
  ROUND( 3 );
  ROUND( 4 );
  ROUND( 5 );
  ROUND( 6 );
  ROUND( 7 );
  ROUND( 8 );
  ROUND( 9 );
  ROUND( 10 );
  ROUND( 11 );
  
  state->h[0] ^= v[0] ^ v[ 8];
  state->h[1] ^= v[1] ^ v[ 9];
  state->h[2] ^= v[2] ^ v[10];
  state->h[3] ^= v[3] ^ v[11];
  state->h[4] ^= v[4] ^ v[12];
  state->h[5] ^= v[5] ^ v[13];
  state->h[6] ^= v[6] ^ v[14];
  state->h[7] ^= v[7] ^ v[15];

  memcpy(hash, (uchar *)state->h, outlen);
}
